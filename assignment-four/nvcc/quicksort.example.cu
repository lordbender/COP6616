#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define MAX_DEPTH       16
#define INSERTION_SORT  32

__device__ void selection_sort(unsigned int *data, int left, int right)
{
    for (int i = left ; i <= right ; ++i)
    {
        unsigned min_val = data[i];
        int min_idx = i;

        // Find the smallest value in the range [left, right].
        for (int j = i+1 ; j <= right ; ++j)
        {
            unsigned val_j = data[j];

            if (val_j < min_val)
            {
                min_idx = j;
                min_val = val_j;
            }
        }

        // Swap the values.
        if (i != min_idx)
        {
            data[min_idx] = data[i];
            data[i] = min_val;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
// Very basic quicksort algorithm, recursively launching the next level.
////////////////////////////////////////////////////////////////////////////////
__global__ void cdp_simple_quicksort(unsigned int *data, int left, int right, int depth)
{
    // If we're too deep or there are few elements left, we use an insertion sort...
    if (depth >= MAX_DEPTH || right-left <= INSERTION_SORT)
    {
        selection_sort(data, left, right);
        return;
    }

    unsigned int *lptr = data+left;
    unsigned int *rptr = data+right;
    unsigned int  pivot = data[(left+right)/2];

    // Do the partitioning.
    while (lptr <= rptr)
    {
        // Find the next left- and right-hand values to swap
        unsigned int lval = *lptr;
        unsigned int rval = *rptr;

        // Move the left pointer as long as the pointed element is smaller than the pivot.
        while (lval < pivot)
        {
            lptr++;
            lval = *lptr;
        }

        // Move the right pointer as long as the pointed element is larger than the pivot.
        while (rval > pivot)
        {
            rptr--;
            rval = *rptr;
        }

        // If the swap points are valid, do the swap!
        if (lptr <= rptr)
        {
            *lptr++ = rval;
            *rptr-- = lval;
        }
    }

    // Now the recursive part
    int nright = rptr - data;
    int nleft  = lptr - data;

    // Launch a new block to sort the left part.
    if (left < (rptr-data))
    {
        hipStream_t s;
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s >>>(data, left, nright, depth+1);
        hipStreamDestroy(s);
    }

    // Launch a new block to sort the right part.
    if ((lptr-data) < right)
    {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s1 >>>(data, nleft, right, depth+1);
        hipStreamDestroy(s1);
    }
}

////////////////////////////////////////////////////////////////////////////////
// Call the quicksort kernel from the host.
////////////////////////////////////////////////////////////////////////////////
void run_qsort(unsigned int *data, unsigned int nitems)
{
    // Prepare CDP for the max depth 'MAX_DEPTH'.
    gpuErrchk(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH));

    // Launch on device
    int left = 0;
    int right = nitems-1;
    std::cout << "Launching kernel on the GPU" << std::endl;
    cdp_simple_quicksort<<< 1, 1 >>>(data, left, right, 0);
    gpuErrchk(hipGetLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipGetLastError());
}

////////////////////////////////////////////////////////////////////////////////
// Initialize data on the host.
////////////////////////////////////////////////////////////////////////////////
void initialize_data(unsigned int *dst, unsigned int nitems)
{
    // Fixed seed for illustration
    srand(2047);

    // Fill dst with random values
    for (unsigned i = 0 ; i < nitems ; i++)
        dst[i] = rand() % nitems ;
}

////////////////////////////////////////////////////////////////////////////////
// Verify the results.
////////////////////////////////////////////////////////////////////////////////
void check_results(int n, unsigned int *results_d)
{
    unsigned int *results_h = new unsigned[n];
    gpuErrchk(hipMemcpy(results_h, results_d, n*sizeof(unsigned), hipMemcpyDeviceToHost));

    for (int i = 1 ; i < n ; ++i)
        printf("\t%d", results_h)

    std::cout << "OK" << std::endl;
    delete[] results_h;
}

////////////////////////////////////////////////////////////////////////////////
// Main entry point.
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    int num_items = 128;
    bool verbose = false;

    // Create input data
    unsigned int *h_data = 0;
    unsigned int *d_data = 0;

    // Allocate CPU memory and initialize data.
    std::cout << "Initializing data:" << std::endl;
    h_data =(unsigned int *)malloc(num_items*sizeof(unsigned int));
    initialize_data(h_data, num_items);

    // Allocate GPU memory.
    gpuErrchk(hipMalloc((void **)&d_data, num_items * sizeof(unsigned int)));
    gpuErrchk(hipMemcpy(d_data, h_data, num_items * sizeof(unsigned int), hipMemcpyHostToDevice));

    // Execute
    std::cout << "Running quicksort on " << num_items << " elements" << std::endl;
    run_qsort(d_data, num_items);

    // Check result
    std::cout << "Validating results: ";
    check_results(num_items, d_data);

    free(h_data);
    gpuErrchk(hipFree(d_data));

    exit(EXIT_SUCCESS);
}

