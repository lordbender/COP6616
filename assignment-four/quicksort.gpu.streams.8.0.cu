
#include <hip/hip_runtime.h>
// #include <stdio.h>
// #include <stdlib.h>
// #include <time.h>

// #include "main_cuda.cuh"

// static const int BLOCK_SIZE = 256;

// __device__ void swap_device(int *a, int *b)
// {
//     int t = *a;
//     *a = *b;
//     *b = t;
// }

// __device__ int partition_device(int *arr, int low, int high)
// {
//     int pivot = arr[high];
//     int i = (low - 1);

//     for (int j = low; j <= high - 1; j++)
//     {
//         if (arr[j] <= pivot)
//         {
//             i++;
//             swap_device(&arr[i], &arr[j]);
//         }
//     }
//     swap_device(&arr[i + 1], &arr[high]);
//     return (i + 1);
// }

// __global__ void quicksort_device(int *data, int left, int right)
// {
//     cudaStream_t s1, s2;

//     int pi = partition_device(data, left, right);

//     int nright = pi - 1;
//     int nleft = pi + 1;

//     if (left < nright)
//     {
//         cudaStreamCreateWithFlags(&s1, cudaStreamNonBlocking);
//         quicksort_device<<<1, 64, 0, s1>>>(data, left, nright);
//     }

//     if (nleft < right)
//     {
//         cudaStreamCreateWithFlags(&s2, cudaStreamNonBlocking);
//         quicksort_device<<<1, 64, 0, s2>>>(data, nleft, right);
//     }

//     return;
// }

// double quicksort_gpu_streams(int size)
// {
//     int *ha, *da;

//     ha = (int *)malloc(sizeof(int) * size);

//     for (int i = 0; i < size; i++)
//     {
//         ha[i] = rand();
//     }

//     clock_t start = clock();

//     gpuErrchk(cudaMalloc((void **)&da, sizeof(int) * size));
//     gpuErrchk(cudaGetLastError());

//     gpuErrchk(cudaMemcpy(da, ha, sizeof(int) * size, cudaMemcpyHostToDevice));
//     gpuErrchk(cudaGetLastError());

//     int grid = ceil(size * 1.0 / BLOCK_SIZE);

//     quicksort_device<<<grid, BLOCK_SIZE>>>(da, 0, size - 1);

// 	cudaStreamSynchronize(0);
//     gpuErrchk(cudaGetLastError());

//     cudaMemcpy(ha, da, sizeof(int) * size, cudaMemcpyDeviceToHost);

//     cudaFree(da);
//     cudaDeviceReset();

//     free(ha);

//     clock_t end = clock();
//     return time_calc(start, end);
// }