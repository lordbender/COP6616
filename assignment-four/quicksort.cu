#include "hip/hip_runtime.h"
#include <stdio.h>
#include "main_cuda.h"

__global__ 
void hello(char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}

void quicksort(int size, int blocksize) {

	char a[size] = "Hello \0\0\0\0\0\0";
	int b[size] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 
	char *ad;
	int *bd;
	const int csize = size*sizeof(char);
	const int isize = size*sizeof(int);
 
	printf("%s", a);
 
	hipMalloc( (void**)&ad, csize ); 
	hipMalloc( (void**)&bd, isize ); 
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
	hipFree( ad );
    hipFree( bd );
    
    printf("%s\n", a);
}