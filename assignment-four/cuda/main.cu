
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define MAX_DEPTH       32
#define INSERTION_SORT  64

__device__ void selection_sort(unsigned int *data, int left, int right)
{
    for (int i = left ; i <= right ; ++i)
    {
        unsigned min_val = data[i];
        int min_idx = i;

        for (int j = i+1 ; j <= right ; ++j)
        {
            unsigned val_j = data[j];

            if (val_j < min_val)
            {
                min_idx = j;
                min_val = val_j;
            }
        }

        if (i != min_idx)
        {
            data[min_idx] = data[i];
            data[i] = min_val;
        }
    }
}

__global__ void quicksort_gpu(unsigned int *data, int left, int right, int depth)
{
    if (depth >= MAX_DEPTH || right-left <= INSERTION_SORT)
    {
        selection_sort(data, left, right);
        return;
    }

    unsigned int *lptr = data+left;
    unsigned int *rptr = data+right;
    unsigned int  pivot = data[(left+right)/2];

    // Do the partitioning.
    while (lptr <= rptr)
    {
        unsigned int lval = *lptr;
        unsigned int rval = *rptr;

        while (lval < pivot)
        {
            lptr++;
            lval = *lptr;
        }

        while (rval > pivot)
        {
            rptr--;
            rval = *rptr;
        }

        if (lptr <= rptr)
        {
            *lptr++ = rval;
            *rptr-- = lval;
        }
    }

    int nright = rptr - data;
    int nleft  = lptr - data;

    if (left < (rptr-data))
    {
        hipStream_t s;
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        quicksort_gpu<<< 1, 1, 0, s >>>(data, left, nright, depth+1);
        hipStreamDestroy(s);
    }

    if ((lptr-data) < right)
    {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        quicksort_gpu<<< 1, 1, 0, s1 >>>(data, nleft, right, depth+1);
        hipStreamDestroy(s1);
    }
}

double run_sort(unsigned int *data, unsigned int size)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int left = 0;
    int right = size-1;
    hipEventRecord(start);
    int grid = ceil(size * 1.0 / 256);
    quicksort_gpu<<< grid, 256 >>>(data, left, right, 0);
    hipEventRecord(stop);
    // gpuErrchk(cudaGetLastError());

    hipDeviceSynchronize();
    
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    return milliseconds;
}

int main(int argc, char **argv)
{
    srand(time(0));

    int size = atoi(argv[1]);

    unsigned int *da = 0;

    unsigned int *ha =(unsigned int *)malloc(size*sizeof(unsigned int));
    for (unsigned i = 0 ; i < size ; i++)
        ha[i] = rand() % size;
       
    hipMalloc((void **)&da, size * sizeof(unsigned int));
    hipMemcpy(da, ha, size * sizeof(unsigned int), hipMemcpyHostToDevice);

    double time = run_sort(da, size);

    unsigned int *results = new unsigned[size];
    hipMemcpy(results, da, size*sizeof(unsigned), hipMemcpyDeviceToHost);
        
    // printf("\n");
    // for (int i = 1 ; i < size ; ++i)
    //     printf("\t%d", results[i]);
    // printf("\n");
    
    hipFree(da);
    hipDeviceReset();
    free(ha);
    delete[] results;
    
    printf("\tCPU O(n*log(n)) GPU Quicksort: Completed %d numbers in %f seconds!!!\n", size, (time / 1000.0));

    exit(EXIT_SUCCESS);
}

