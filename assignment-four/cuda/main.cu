
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <ctime>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void quicksort_gpu(unsigned int *data, int left, int right)
{
    unsigned int *lptr = data+left;
    unsigned int *rptr = data+right;
    unsigned int  pivot = data[(left+right)/2];

    // Do the partitioning.
    while (lptr <= rptr)
    {
        unsigned int lval = *lptr;
        unsigned int rval = *rptr;

        while (lval < pivot)
        {
            lptr++;
            lval = *lptr;
        }

        while (rval > pivot)
        {
            rptr--;
            rval = *rptr;
        }

        if (lptr <= rptr)
        {
            *lptr++ = rval;
            *rptr-- = lval;
        }
    }

    int nright = rptr - data;
    int nleft  = lptr - data;

    if (left < (rptr-data))
    {
        hipStream_t s;
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        quicksort_gpu<<< 1, 1, 0, s >>>(data, left, nright);
        hipStreamDestroy(s);
    }

    if ((lptr-data) < right)
    {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        quicksort_gpu<<< 1, 1, 0, s1 >>>(data, nleft, right);
        hipStreamDestroy(s1);
    }
}

int main(int argc, char **argv)
{
    srand(time(0));
    int size = atoi(argv[1]);

    unsigned int *ha =(unsigned int *)malloc(size*sizeof(unsigned int));
    unsigned int *da = 0;

    for (unsigned i = 0 ; i < size ; i++)
        ha[i] = rand() % size;

    std::clock_t start = std::clock();  

    hipMalloc((void **)&da, size * sizeof(unsigned int));
    hipMemcpy(da, ha, size * sizeof(unsigned int), hipMemcpyHostToDevice);

    int left = 0;
    int right = size-1;

    quicksort_gpu<<< 1, 32 >>>(da, left, right);
    gpuErrchk(hipGetLastError());
    hipDeviceSynchronize();

    unsigned int *results = new unsigned[size];
    hipMemcpy(results, da, size*sizeof(unsigned), hipMemcpyDeviceToHost);
        
    double duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;

    // printf("\n");
    // for (int i = 1 ; i < size ; ++i)
    //     printf("\t%d", results[i]);
    // printf("\n");
    
    hipFree(da);
    hipDeviceReset();
    free(ha);
    delete[] results;
    
    printf("\tGPU O(n*log(n)) GPU Quicksort: Completed %d numbers in %f seconds!!!\n", size, duration);

    exit(EXIT_SUCCESS);
}

