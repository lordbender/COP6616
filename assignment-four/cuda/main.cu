
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define MAX_DEPTH       16
#define INSERTION_SORT  32

__device__ void selection_sort(unsigned int *data, int left, int right)
{
    for (int i = left ; i <= right ; ++i)
    {
        unsigned min_val = data[i];
        int min_idx = i;

        for (int j = i+1 ; j <= right ; ++j)
        {
            unsigned val_j = data[j];

            if (val_j < min_val)
            {
                min_idx = j;
                min_val = val_j;
            }
        }

        if (i != min_idx)
        {
            data[min_idx] = data[i];
            data[i] = min_val;
        }
    }
}

__global__ void quicksort_gpu(unsigned int *data, int left, int right, int depth)
{
    if (depth >= MAX_DEPTH || right-left <= INSERTION_SORT)
    {
        selection_sort(data, left, right);
        return;
    }

    unsigned int *lptr = data+left;
    unsigned int *rptr = data+right;
    unsigned int  pivot = data[(left+right)/2];

    // Do the partitioning.
    while (lptr <= rptr)
    {
        unsigned int lval = *lptr;
        unsigned int rval = *rptr;

        while (lval < pivot)
        {
            lptr++;
            lval = *lptr;
        }

        while (rval > pivot)
        {
            rptr--;
            rval = *rptr;
        }

        if (lptr <= rptr)
        {
            *lptr++ = rval;
            *rptr-- = lval;
        }
    }

    int nright = rptr - data;
    int nleft  = lptr - data;

    if (left < (rptr-data))
    {
        hipStream_t s;
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        quicksort_gpu<<< 1, 1, 0, s >>>(data, left, nright, depth+1);
        hipStreamDestroy(s);
    }

    if ((lptr-data) < right)
    {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        quicksort_gpu<<< 1, 1, 0, s1 >>>(data, nleft, right, depth+1);
        hipStreamDestroy(s1);
    }
}

void run_sort(unsigned int *data, unsigned int size)
{
    gpuErrchk(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH));

    int left = 0;
    int right = size-1;

    quicksort_gpu<<< 1, 1 >>>(data, left, right, 0);
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipGetLastError());
}

int main(int argc, char **argv)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    srand(time(0));

    int size = atoi(argv[1]);

    unsigned int *da = 0;

    unsigned int *ha =(unsigned int *)malloc(size*sizeof(unsigned int));
    for (unsigned i = 0 ; i < size ; i++)
        ha[i] = rand() % size;
       
    hipEventRecord(start);
    gpuErrchk(hipMalloc((void **)&da, size * sizeof(unsigned int)));
    gpuErrchk(hipMemcpy(da, ha, size * sizeof(unsigned int), hipMemcpyHostToDevice));

    run_sort(da, size);

    unsigned int *results = new unsigned[size];
    gpuErrchk(hipMemcpy(results, da, size*sizeof(unsigned), hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    
    // printf("\n");
    // for (int i = 1 ; i < size ; ++i)
    //     printf("\t%d", results[i]);
    // printf("\n");
    
    gpuErrchk(hipFree(da));
    free(ha);
    delete[] results;
    
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("\tCPU O(n*log(n)) GPU Quicksort: Completed %d numbers in %f seconds!!!\n", size, milliseconds);

    exit(EXIT_SUCCESS);
}

