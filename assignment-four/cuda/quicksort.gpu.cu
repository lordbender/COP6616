
#include <hip/hip_runtime.h>
// #include <stdio.h>

// #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
// inline void gpuAssert(cudaError_t code, const char *file, int line, bool abort=true)
// {
//    if (code != cudaSuccess) 
//    {
//       fprintf(stderr,"GPUassert: %s %s %d\n", cudaGetErrorString(code), file, line);
//       if (abort) exit(code);
//    }
// }

// static const int BLOCK_SIZE = 256;

// __device__ void swap_device(int *a, int *b)
// {
//     int t = *a;
//     *a = *b;
//     *b = t;
// }

// __device__ int partition_device(int *arr, int low, int high)
// {
//     int pivot = arr[high];
//     int i = (low - 1);

//     for (int j = low; j <= high - 1; j++)
//     {
//         if (arr[j] <= pivot)
//         {
//             i++;
//             swap_device(&arr[i], &arr[j]);
//         }
//     }
//     swap_device(&arr[i + 1], &arr[high]);
//     return (i + 1);
// }

// // Based on CUDA Examples - But Optimized
// __global__ void quicksort_device(int *data, int left, int right)
// { 
//     cudaStream_t s1, s2;
//     int pi = partition_device(data, left, right);

//     int nright = pi - 1;
//     int nleft = pi + 1;

//     if (left < nright)
//     {
//         cudaStreamCreateWithFlags(&s1, cudaStreamNonBlocking);
//         quicksort_device<<<1, 32, 0, s1>>>(data, left, nright);
//     }

//     if (nleft < right)
//     {
//         cudaStreamCreateWithFlags(&s2, cudaStreamNonBlocking);
//         quicksort_device<<<1, 32, 0, s2>>>(data, nleft, right);
//     }
// }

// void quicksort_host(int *da, int *hc, int size)
// {
//     // Call the device.
//     int grid = ceil(size * 1.0 / BLOCK_SIZE);
//     quicksort_device<<<grid, BLOCK_SIZE>>>(da, 0, size - 1);
//     gpuErrchk(cudaGetLastError());

//     // Ensure the Device is in sync, before we copy the data back!
//     gpuErrchk(cudaDeviceSynchronize());
//     gpuErrchk(cudaGetLastError());

//     // Copy the results back from the device.
//     gpuErrchk(cudaMemcpy(hc, da, sizeof(int) * size, cudaMemcpyDeviceToHost));
//     gpuErrchk(cudaGetLastError());

//     // Testing that sort is working, keep commented out on large values of N (say N > 1000)
//     // for (int i = 0; i < size; i++)
//     // {
//     //     printf("\t %d\n", hc[i]);
//     // }
// }

// int main(int argc, char *argv[])
// {
//     cudaEvent_t start, stop;
//     cudaEventCreate(&start);
//     cudaEventCreate(&stop);

//   	srand(time(0));

//  	int size = atoi(argv[1]);

//     int* ha = (int *)malloc(sizeof(int) * size);
//     int* hc = (int *)malloc(sizeof(int) * size);

//     for (int i = 0; i < size; i++)
//     {
//         ha[i] = rand();
//         hc[i] = 0;
//     }

//     int *da;
//     gpuErrchk(cudaMalloc((void **)&da, sizeof(int) * size));
//     gpuErrchk(cudaMemcpy(da, ha, sizeof(int) * size, cudaMemcpyHostToDevice));
//     gpuErrchk(cudaGetLastError());

//     // Kick off the sort!
//     quicksort_host(da, hc, size);

//     gpuErrchk(cudaFree(da));
//     free(ha);
//     free(hc);
    
//     cudaEventSynchronize(stop);
//     float milliseconds = 0;
//     cudaEventElapsedTime(&milliseconds, start, stop);

// 	printf("\tGPU O(n*log(n)) Streamed: Completed %d numbers in %f seconds!!!\n\n", size, milliseconds);
// }
