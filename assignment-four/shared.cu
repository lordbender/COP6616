
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.cuh"

double time_calc(clock_t start, clock_t end)
{
    return ((double)(end - start)) / CLOCKS_PER_SEC;
}

void swap(int array[], int left, int right)
{
	int temp;
	temp = array[left];
	array[left] = array[right];
	array[right] = temp;
}

int partition(int array[], int left, int right, int pivot_index)
{
	int pivot_value = array[pivot_index];
	int store_index = left;
	int i;

	swap(array, pivot_index, right);
	for (i = left; i < right; i++)
		if (array[i] <= pivot_value) {
			swap(array, i, store_index);
			++store_index;
		}
	swap(array, store_index, right);
	return store_index;
}

