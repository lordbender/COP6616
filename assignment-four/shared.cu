
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>
#include "main_cuda.cuh"

using namespace std::chrono;

duration<double> time_calc(high_resolution_clock::time_point start, high_resolution_clock::time_point end)
{
    return duration_cast<duration<double>>(end - start);
}

int getMax(int *arr, int n) 
{ 
    int mx = arr[0]; 
    for (int i = 1; i < n; i++) 
        if (arr[i] > mx) 
            mx = arr[i]; 
    return mx; 
} 

void swap(int array[], int left, int right)
{
	int temp;
	temp = array[left];
	array[left] = array[right];
	array[right] = temp;
}

int partition(int array[], int left, int right, int pivot_index)
{
	int pivot_value = array[pivot_index];
	int store_index = left;
	int i;

	swap(array, pivot_index, right);
	for (i = left; i < right; i++)
		if (array[i] <= pivot_value) {
			swap(array, i, store_index);
			++store_index;
		}
	swap(array, store_index, right);
	return store_index;
}

