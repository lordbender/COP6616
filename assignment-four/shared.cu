
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.cuh"

double time_calc(clock_t start, clock_t end)
{
    return ((double)(end - start)) / CLOCKS_PER_SEC;
}
