#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "main_cuda.cuh"

const int N = 1 << 20;

__global__ void kernel(int *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

int stream_support_test()
{
    const int size = 8;

    hipStream_t streams[size];
    int *da = (int*) malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++) {
        gpuErrchk(hipStreamCreate(&streams[i]));
        gpuErrchk(hipGetLastError());

        gpuErrchk(hipMalloc((void **)&da, sizeof(int) * size));
        gpuErrchk(hipGetLastError());

        // launch one worker kernel per stream
        kernel<<<1, 64, 0, streams[i]>>>(da, N);
        gpuErrchk(hipGetLastError());

        // launch a dummy kernel on the default stream
        kernel<<<1, 1>>>(0, 0);
        gpuErrchk(hipGetLastError());
    }

    hipDeviceReset();

    for (int i = 0; i < size; i++){
        printf("data point %d", da[i]);
    }

    return 0;
}