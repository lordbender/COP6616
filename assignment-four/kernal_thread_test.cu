#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream> 
using namespace std; 

#include "main_cuda.cuh"

const int N = 1 << 20;

__global__ void kernel(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

int stream_support_test()
{
    const int num_streams = 8;

    hipStream_t streams[num_streams];
    float *data[num_streams];

    for (int i = 0; i < num_streams; i++) {
        gpuErrchk(hipStreamCreate(&streams[i]));
        gpuErrchk(hipGetLastError());

        gpuErrchk(hipMalloc(&data[i], N * sizeof(float)));
        gpuErrchk(hipGetLastError());

        // launch one worker kernel per stream
        kernel<<<1, 64, 0, streams[i]>>>(data[i], N);
        gpuErrchk(hipGetLastError());

        // launch a dummy kernel on the default stream
        kernel<<<1, 1>>>(0, 0);
    }

    hipDeviceReset();

    for (int i = 0; i < num_streams; i++) {
        cout << data[i] << endl; 
    }

    return 0;
}