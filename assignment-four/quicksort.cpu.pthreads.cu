#include "hip/hip_runtime.h"
#include <iostream>
#include <thread> 

#include "main_cuda.cuh"

using namespace std; 

void foo(int Z) 
{ 
    for (int i = 0; i < Z; i++) { 
        cout << "Thread using function"
               " pointer as callable\n"; 
    } 
} 

class thread_obj { 
public: 
    void operator()(int x) 
    { 
        for (int i = 0; i < x; i++) 
            cout << "Thread using function"
                  " object as  callable\n"; 
    } 
}; 

double quicksort_cpu_pthreads(int size)
{   
    int *a = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        a[i] = rand();
    }

    clock_t start = clock();

    thread th1(foo, 3); 
    thread th2(thread_obj(), 3); 
  
    // Define a Lambda Expression 
    auto f = [](int x) { 
        for (int i = 0; i < x; i++) 
            cout << "Thread using lambda"
             " expression as callable\n"; 
    }; 
  
    thread th3(f, 3); 
  
    th1.join(); 
    th2.join(); 
    th3.join();
    
    clock_t end = clock();

    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    // for (int i = 0; i < size; i++)
    // {
    //     printf("\t %d\n", a[i]);
    // }

    return time_calc(start, end);
}
