#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>
#include <iostream>
#include <cstdio>

#include "main_cuda.cuh"

#define MAX_DEPTH       16
#define INSERTION_SORT  32

__device__ void selection_sort(unsigned int *data, int left, int right)
{
    for (int i = left ; i <= right ; ++i)
    {
        unsigned min_val = data[i];
        int min_idx = i;

        // Find the smallest value in the range [left, right].
        for (int j = i+1 ; j <= right ; ++j)
        {
            unsigned val_j = data[j];

            if (val_j < min_val)
            {
                min_idx = j;
                min_val = val_j;
            }
        }

        // Swap the values.
        if (i != min_idx)
        {
            data[min_idx] = data[i];
            data[i] = min_val;
        }
    }
}

__global__ void cdp_simple_quicksort(unsigned int *data, int left, int right, int depth)
{
    if (depth >= MAX_DEPTH || right-left <= INSERTION_SORT)
    {
        selection_sort(data, left, right);
        return;
    }

    unsigned int *lptr = data+left;
    unsigned int *rptr = data+right;
    unsigned int  pivot = data[(left+right)/2];

    while (lptr <= rptr)
    {
        unsigned int lval = *lptr;
        unsigned int rval = *rptr;

        while (lval < pivot)
        {
            lptr++;
            lval = *lptr;
        }

        // Move the right pointer as long as the pointed element is larger than the pivot.
        while (rval > pivot)
        {
            rptr--;
            rval = *rptr;
        }

        // If the swap points are valid, do the swap!
        if (lptr <= rptr)
        {
            *lptr++ = rval;
            *rptr-- = lval;
        }
    }

    // Now the recursive part
    int nright = rptr - data;
    int nleft  = lptr - data;

    // Launch a new block to sort the left part.
    if (left < (rptr-data))
    {
        hipStream_t s;
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s >>>(data, left, nright, depth+1);
        hipStreamDestroy(s);
    }

    // Launch a new block to sort the right part.
    if ((lptr-data) < right)
    {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s1 >>>(data, nleft, right, depth+1);
        hipStreamDestroy(s1);
    }
}

////////////////////////////////////////////////////////////////////////////////
// Call the quicksort kernel from the host.
////////////////////////////////////////////////////////////////////////////////
void run_qsort(unsigned int *data, unsigned int nitems)
{
    // Prepare CDP for the max depth 'MAX_DEPTH'.
    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH);

    // Launch on device
    int left = 0;
    int right = nitems-1;
    std::cout << "Launching kernel on the GPU" << std::endl;
    cdp_simple_quicksort<<< 1, 1 >>>(data, left, right, 0);
    hipDeviceSynchronize();
}

////////////////////////////////////////////////////////////////////////////////
// Initialize data on the host.
////////////////////////////////////////////////////////////////////////////////
void initialize_data(unsigned int *dst, unsigned int nitems)
{
    // Fixed seed for illustration
    srand(2047);

    // Fill dst with random values
    for (unsigned i = 0 ; i < nitems ; i++)
        dst[i] = rand() % nitems ;
}


void check_results(int n, unsigned int *results_d)
{
    unsigned int *results_h = new unsigned[n];
    hipMemcpy(results_h, results_d, n*sizeof(unsigned), hipMemcpyDeviceToHost);


    delete[] results_h;
}

duration<double> quicksort_gpu_streams(int size)
{
    unsigned int *h_data = 0;
    unsigned int *d_data = 0;

    h_data =(unsigned int *)malloc(size*sizeof(unsigned int));
    initialize_data(h_data, size);

    high_resolution_clock::time_point start = high_resolution_clock::now();
    hipMalloc((void **)&d_data, size * sizeof(unsigned int));
    hipMemcpy(d_data, h_data, size * sizeof(unsigned int), hipMemcpyHostToDevice);

    run_qsort(d_data, size);

    check_results(size, d_data);

    for (int i = 0; i < size; i++)
    {
        printf("\t d_data[ %d ] => %d\n", i, d_data[i]);
    }

    free(h_data);
    hipFree(d_data);
    high_resolution_clock::time_point end = high_resolution_clock::now();

    return time_calc(start, end);
}

