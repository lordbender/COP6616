#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.cuh"

void quicksort(int arr[], int low, int high) 
{ 
	int pivot_index = low;

	if (low < high) 
	{ 
		int pi = partition(arr, low, high, pivot_index); 

		quicksort(arr, low, pi - 1); 
		quicksort(arr, pi + 1, high); 
	} 
} 

double quicksort_cpu(int size) {
	int *ha = (int*)malloc(sizeof(int) * size);
  
	for (int i = 0; i < size; i++)
	{
		ha[i] = rand();
	}

	clock_t start = clock();
	quicksort(ha, 0, size - 1);
	clock_t end= clock();

	// Testing that sort is working, keep commented out on large values of N (say N > 1000)
	// for (int i = 0; i < size; i++) {
    // 	printf("\t %d\n", ha_seq[i]);
	// }

	return time_calc(start, end);
}