#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.cuh"
 
void swap(int* a, int* b) 
{ 
	int t = *a; 
	*a = *b; 
	*b = t; 
} 

int partition (int arr[], int low, int high) 
{ 
	int pivot = arr[high]; // pivot 
	int i = (low - 1); // Index of smaller element 

	for (int j = low; j <= high- 1; j++) 
	{ 
		if (arr[j] <= pivot) 
		{ 
			i++;
			swap(&arr[i], &arr[j]); 
		} 
	} 
	swap(&arr[i + 1], &arr[high]); 
	return (i + 1); 
} 

void quicksort(int arr[], int low, int high) 
{ 
	if (low < high) 
	{ 
		int pi = partition(arr, low, high); 

		quicksort(arr, low, pi - 1); 
		quicksort(arr, pi + 1, high); 
	} 
} 

double quicksort_cpu(int size) {
	int *ha = (int*)malloc(sizeof(int) * size);
  
	for (int i = 0; i < size; i++)
	{
		ha[i] = rand();
	}

	clock_t start = clock();
	quicksort(ha, 0, size - 1);
	clock_t end= clock();

	// Testing that sort is working, keep commented out on large values of N (say N > 1000)
	// for (int i = 0; i < size; i++) {
    // 	printf("\t %d\n", ha_seq[i]);
	// }

	return time_calc(start, end);
}