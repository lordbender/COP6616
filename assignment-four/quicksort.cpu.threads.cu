// #define _XOPEN_SOURCE 600
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <pthread.h>

#include "main_cuda.cuh"

struct qsort_starter
{
	int *array;
	int left;
	int right;
	int depth;
};

// Macro for swapping two values.
#define SWAP(x,y) do {\
    __typeof__(x) tmp = x;\
    x = y;\
    y = tmp;\
} while(0)

void quicksort_serial(int *array, int left, int right)
{
     if (right > left)
     {
        int pivotIndex = left + (right - left)/2;
        pivotIndex = partition(array, left, right, pivotIndex);
        quicksort_serial(array, left, pivotIndex-1);
        quicksort_serial(array, pivotIndex+1, right);
     }
}


void parallel_quicksort(int *array, int left, int right, int depth);

void* quicksort_thread(void *init)
{
    struct qsort_starter *start = (qsort_starter*)init;
    parallel_quicksort(start->array, start->left, start->right, start->depth);
    return NULL;
}

void parallel_quicksort(int *array, int left, int right, int depth)
{
    if (right > left)
    {
        int pivotIndex = left + (right - left)/2;
        pivotIndex = partition(array, left, right, pivotIndex);
        // Either do the parallel or serial quicksort, depending on the depth
        // specified.
        if (depth-- > 0)
        {
            // Create the thread for the first recursive call
            struct qsort_starter arg = {array, left, pivotIndex-1, depth};
            pthread_t thread;
            int ret = pthread_create(&thread, NULL, quicksort_thread, &arg);
            assert((ret == 0) && "Thread creation failed");
            // Perform the second recursive call in this thread
            parallel_quicksort(array, pivotIndex+1, right, depth);
            // Wait for the first call to finish.
            pthread_join(thread, NULL);
        }
        else
        {
            quicksort_serial(array, left, pivotIndex-1);
            quicksort_serial(array, pivotIndex+1, right);
        }
    }
}


duration<double> quicksort_cpu_threads(int size)
{   
	int depth = 5;

    int *a = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        a[i] = rand();
    }
 
  	high_resolution_clock::time_point start = high_resolution_clock::now();
	parallel_quicksort(a, 0, size - 1, depth); 
    high_resolution_clock::time_point end = high_resolution_clock::now();
	
    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    for (int i = 0; i < size; i++)
    {
        printf("\t %d\n", a[i]);
    }

    return time_calc(start, end);
}
