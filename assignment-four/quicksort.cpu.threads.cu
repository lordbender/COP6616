#include "hip/hip_runtime.h"
// https://www.geeksforgeeks.org/multithreading-in-cpp/
// https://github.com/markwkm/quicksort/blob/master/recursive/quicksort-parallel.c

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/wait.h>
#include <sys/ipc.h>
#include <sys/shm.h>

#include "main_cuda.cuh"

using namespace std; 

void quicksort_threaded(int *array, int left, int right)
{
	int pivot_index = left;
	int pivot_new_index;

	/*
	 * Use -1 to initialize because fork() uses 0 to identify a process as a
	 * child.
	 */
	int lchild = -1;
	int rchild = -1;

	if (right > left) {
		int status; /* For waitpid() only. */

		pivot_new_index = partition(array, left, right, pivot_index);

		/*
		 * Parallize by processing the left and right partion siultaneously.
		 * Start by spawning the 'left' child.
		 */
		lchild = fork();
		if (lchild < 0) {
			perror("fork");
			exit(1);
		}
		if (lchild == 0) {
			/* The 'left' child starts processing. */
			quicksort_threaded(array, left, pivot_new_index - 1);
			exit(0);
		} else {
			/* The parent spawns the 'right' child. */
			rchild = fork();
			if (rchild < 0) {
				perror("fork");
				exit(1);
			}
			if (rchild == 0) {
				/* The 'right' child starts processing. */
				quicksort_threaded(array, pivot_new_index + 1, right);
				exit(0);
			}
		}
		/* Parent waits for children to finish. */
		waitpid(lchild, &status, 0);
		waitpid(rchild, &status, 0);
	}
}

double quicksort_cpu_threads(int size)
{   
	size = size < 5000 ? size : 5000;
	
    int *a = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        a[i] = rand();
    }

    clock_t start = clock();

    quicksort_threaded(a, 0, size - 1); 
    
    clock_t end = clock();

    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    // for (int i = 0; i < size; i++)
    // {
    //     printf("\t %d\n", a[i]);
    // }

    return time_calc(start, end);
}
