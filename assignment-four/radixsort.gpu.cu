#include "hip/hip_runtime.h"
// http://on-demand.gputechconf.com/gtc/2014/presentations/S4158-cuda-streams-best-practices-common-pitfalls.pdf
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <vector>
#include <ratio>
#include <chrono>

#include "main_cuda.cuh"

using namespace std::chrono;

static const int BLOCK_SIZE = 256;

__global__ void countsort_device(int *arr, int n, int exp) 
{ 
    int *output = (int *)malloc(sizeof(int) * n); 
    int i, count[10] = {0}; 
  
    for (i = 0; i < n; i++) 
        count[ (arr[i]/exp)%10 ]++; 

    for (i = 1; i < 10; i++) 
        count[i] += count[i - 1]; 
  
    // Build the output array 
    for (i = n - 1; i >= 0; i--) 
    { 
        output[count[ (arr[i]/exp)%10 ] - 1] = arr[i]; 
        count[ (arr[i]/exp)%10 ]--; 
    } 

    for (i = 0; i < n; i++) 
        arr[i] = output[i]; 
} 

void radixsort_host(int *ha, int size) 
{ 
    int *hc, *da;
    hc = (int *)malloc(sizeof(int) * size);

    int m = getMax(ha, size); 

    hipStream_t *streams = (hipStream_t *)malloc(sizeof(hipStream_t) * size);

    gpuErrchk(hipMalloc((void **)&da, sizeof(int) * size));
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice));
    gpuErrchk(hipGetLastError());

    int grid = ceil(size * 1.0 / BLOCK_SIZE);

    int i = 0;
    for (int exp = 1; m/exp > 0; exp *= 10) {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        streams[i++]= s1;
    }

    i = 0;
    for (int exp = 1; m/exp > 0; exp *= 10) {
        countsort_device<<<grid, BLOCK_SIZE, 0, streams[i++]>>>(ha, size, exp);
    }

    hipStreamSynchronize(0);
    gpuErrchk(hipGetLastError());

    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
    
    gpuErrchk(hipMemcpy(hc, da, sizeof(int) * size, hipMemcpyDeviceToHost));
    gpuErrchk(hipGetLastError());

    hipFree(da);
    hipDeviceReset();
    
    for (int i = 0; i < size; i++)
        ha[i] = hc[i];

    free(hc);
    free(streams);
}


duration<double> radixsort_gpu(int size)
{
   int *ha = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
        ha[i] = rand();

    high_resolution_clock::time_point start = high_resolution_clock::now();
    radixsort_host(ha, size);
    high_resolution_clock::time_point end = high_resolution_clock::now();

    
    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    // for (int i = 0; i < size; i++)
    // {
        //     printf("\t %d\n", ha[i]);
        // }
        
    free(ha);
    return time_calc(start, end);
}
