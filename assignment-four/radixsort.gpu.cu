#include "hip/hip_runtime.h"
// http://on-demand.gputechconf.com/gtc/2014/presentations/S4158-cuda-streams-best-practices-common-pitfalls.pdf
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>

#include "main_cuda.cuh"

using namespace std::chrono;

static const int BLOCK_SIZE = 256;

__global__ void radixsort_device(int *array, int left, int right) {
    // do the work!!!

    return;
}

void radixsort_host(int *arr, int n) 
{ 
    int m = getMax(arr, n); 
  
    for (int exp = 1; m/exp > 0; exp *= 10) 
     {
         // Do work
     }
}

duration<double> radixsort_gpu(int size)
{
    int *ha  = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
    }

    high_resolution_clock::time_point start = high_resolution_clock::now();
    radixsort_host(ha, size);    
    high_resolution_clock::time_point end = high_resolution_clock::now();

    free(ha);

    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    // for (int i = 0; i < size; i++)
    // {
    //     printf("\t %d\n", ha[i]);
    // }

    return time_calc(start, end);
}
