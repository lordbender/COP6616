#include "hip/hip_runtime.h"
// http://on-demand.gputechconf.com/gtc/2014/presentations/S4158-cuda-streams-best-practices-common-pitfalls.pdf
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <vector>
#include <ratio>
#include <chrono>

#include "main_cuda.cuh"

using namespace std::chrono;

static const int BLOCK_SIZE = 256;

__global__ void countsort_device(int *arr, int *output, int n, int exp)
{
    // int id = blockIdx.x * blockDim.x + threadIdx.x;
    // if (id < n)
    //     c[id] = a[id] * a[id];

    int i, count[10] = {0};

    for (i = 0; i < n; i++)
        count[(arr[i] / exp) % 10]++;

    for (i = 1; i < 10; i++)
        count[i] += count[i - 1];

    // Build the output array
    for (i = n - 1; i >= 0; i--)
    {
        output[count[(arr[i] / exp) % 10] - 1] = arr[i];
        count[(arr[i] / exp) % 10]--;
    }
}

void radixsort_host(int *ha, int *hc, int n)
{
    int *da, *dc;

    gpuErrchk(hipMalloc((void **)&da, sizeof(int) * n));
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMalloc((void **)&dc, sizeof(int) * n));
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(da, ha, sizeof(int) * n, hipMemcpyHostToDevice));
    gpuErrchk(hipGetLastError());

    int m = getMax(ha, n);
    int grid = ceil(n * 1.0 / BLOCK_SIZE);

    for (int exp = 1; m / exp > 0; exp *= 10)
    {
        countsort_device<<<grid, BLOCK_SIZE>>>(da, dc, n, exp);
    }

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(hc, dc, sizeof(int) * n, hipMemcpyDeviceToHost));
    gpuErrchk(hipGetLastError());

    hipFree(da);
    hipFree(dc);

    hipDeviceReset();
}

duration<double> radixsort_gpu(int size)
{
    int *ha = (int *)malloc(sizeof(int) * size);
    int *hc = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
        hc[i] = 0;
    }

    high_resolution_clock::time_point start = high_resolution_clock::now();
    radixsort_host(ha, hc, size);
    high_resolution_clock::time_point end = high_resolution_clock::now();

    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    for (int i = 0; i < size; i++)
    {
        printf("\t %d\n", hc[i]);
    }

    free(ha);
    free(hc);

    return time_calc(start, end);
}
