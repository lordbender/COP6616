#include "hip/hip_runtime.h"
// http://on-demand.gputechconf.com/gtc/2014/presentations/S4158-cuda-streams-best-practices-common-pitfalls.pdf
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <vector>
#include <ratio>
#include <chrono>

#include "main_cuda.cuh"

using namespace std::chrono;

static const int BLOCK_SIZE = 256;

__global__ void countsort_device(int *arr, int *c, int n, int exp) 
{ 
    int *output = (int *)malloc(sizeof(int) * n); 
    int i, count[10] = {0}; 
  
    for (i = 0; i < n; i++) 
        count[ (arr[i]/exp)%10 ]++; 

    for (i = 1; i < 10; i++) 
        count[i] += count[i - 1]; 
  
    // Build the output array 
    for (i = n - 1; i >= 0; i--) 
    { 
        output[count[ (arr[i]/exp)%10 ] - 1] = arr[i]; 
        count[ (arr[i]/exp)%10 ]--; 
    } 

    for (i = 0; i < n; i++) 
        c[i] = output[i]; 
} 

void radixsort_host(int *ha, int *hc, int size) 
{ 
    int *da, *dc;

    gpuErrchk(hipMalloc((void **)&da, sizeof(int) * size));
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMalloc((void **)&dc, sizeof(int) * size));
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice));
    gpuErrchk(hipGetLastError());

    int m = getMax(ha, size); 
    hipStream_t streams[size];

    int grid = ceil(size * 1.0 / BLOCK_SIZE);

    int i = 0;
    for (int exp = 1; m/exp > 0; exp *= 10) {
        hipStreamCreate(&streams[i]);  

        gpuErrchk(hipMalloc((void **)&ha, sizeof(int) * size));
        gpuErrchk(hipGetLastError());
        
        countsort_device<<<grid, BLOCK_SIZE, 0, streams[i++]>>>(da, dc, size, exp);
    }

    hipStreamSynchronize(0);
    hipMemcpy(hc, dc, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(dc);
    hipDeviceReset();
}


duration<double> radixsort_gpu(int size)
{
   int *ha = (int *)malloc(sizeof(int) * size);
   int *hc = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++) {
        ha[i] = rand();
        hc[i] = 0;
    }

    high_resolution_clock::time_point start = high_resolution_clock::now();
    radixsort_host(ha, hc, size);
    high_resolution_clock::time_point end = high_resolution_clock::now();

    
    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    for (int i = 0; i < size; i++)
    {
      printf("\t %d\n", hc[i]);
    }
        
    free(ha);
    free(hc);

    return time_calc(start, end);
}
