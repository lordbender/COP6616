#include "hip/hip_runtime.h"
// http://on-demand.gputechconf.com/gtc/2014/presentations/S4158-cuda-streams-best-practices-common-pitfalls.pdf
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <vector>
#include <ratio>
#include <chrono>

#include "main_cuda.cuh"

using namespace std::chrono;

static const int BLOCK_SIZE = 256;

__global__ void countsort_device(int *arr, int n, int exp) 
{ 
    int *output = (int *)malloc(sizeof(int) * n); 
    int i, count[10] = {0}; 
  
    for (i = 0; i < n; i++) 
        count[ (arr[i]/exp)%10 ]++; 

    for (i = 1; i < 10; i++) 
        count[i] += count[i - 1]; 
  
    // Build the output array 
    for (i = n - 1; i >= 0; i--) 
    { 
        output[count[ (arr[i]/exp)%10 ] - 1] = arr[i]; 
        count[ (arr[i]/exp)%10 ]--; 
    } 

    for (i = 0; i < n; i++) 
        arr[i] = output[i]; 
} 

void radixsort_host(int *ha, int size) 
{ 
    int *hc, *da;
    hc = (int *)malloc(sizeof(int) * size);

    int m = getMax(ha, size); 
    std::vector< hipStream_t > streams;

    gpuErrchk(hipMalloc((void **)&da, sizeof(int) * size));
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice));
    gpuErrchk(hipGetLastError());

    for (int exp = 1; m/exp > 0; exp *= 10) {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        streams.push_back(s1);

        int grid = ceil(size * 1.0 / BLOCK_SIZE);
        countsort_device<<<grid, BLOCK_SIZE, 0, s1>>>(ha, size, exp);
    }

    hipStreamSynchronize(0);
    gpuErrchk(hipGetLastError());

    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    hipMemcpy(hc, da, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(da);
    hipDeviceReset();
    
    for (int i = 0; i < size; i++)
        ha[i] = hc[i];

    free(ha);
    free(hc);
}


duration<double> radixsort_gpu(int size)
{
   int *ha = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
        ha[i] = rand();

    high_resolution_clock::time_point start = high_resolution_clock::now();
    radixsort_host(ha, size);
    high_resolution_clock::time_point end = high_resolution_clock::now();

    free(ha);

    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    // for (int i = 0; i < size; i++)
    // {
    //     printf("\t %d\n", ha[i]);
    // }

    return time_calc(start, end);
}
