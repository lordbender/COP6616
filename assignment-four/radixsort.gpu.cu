#include "hip/hip_runtime.h"
// http://on-demand.gputechconf.com/gtc/2014/presentations/S4158-cuda-streams-best-practices-common-pitfalls.pdf
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>

#include "main_cuda.cuh"

using namespace std::chrono;

static const int BLOCK_SIZE = 256;

__device__ void partition_by_bit(int *values, int bit);

__global__ void radix_sort(int *values)
{
    int  bit;
    for( bit = 0; bit < 32; ++bit )
    {
        partition_by_bit(values, bit);
        __syncthreads();
    }
}

template<class T>
__device__ T plus_scan(T *x)
{
    int i = threadIdx.x; // id of thread executing this instance
    int n = blockDim.x;  // total number of threads in this block
    int offset;          // distance between elements to be added

    for( offset = 1; offset < n; offset *= 2) {
        T t;

        if ( i >= offset ) 
            t = x[i-offset];
        
        __syncthreads();

        if ( i >= offset ) 
            x[i] = t + x[i];   

        __syncthreads();
    }
    return x[i];
}


__device__ void partition_by_bit(int *values, int bit)
{
    int i = threadIdx.x;
    int size = blockDim.x;
    int x_i = values[i];          // value of integer at position i
    int p_i = (x_i >> bit) & 1;   // value of bit at position bit

    values[i] = p_i;  

    __syncthreads();

    int T_before = plus_scan(values);

    int T_total  = values[size-1];

    int F_total  = size - T_total;

    __syncthreads();

    if ( p_i )
        values[T_before-1 + F_total] = x_i;
    else
        values[i - T_before] = x_i;
}

duration<double> radixsort_gpu(int size)
{
    int *ha, *hc, *da, *dc;

    ha = (int *)malloc(sizeof(int) * size);
    hc = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
        hc[i] = 0;
    }

    high_resolution_clock::time_point start = high_resolution_clock::now();

    gpuErrchk(hipMalloc((void **)&da, sizeof(int) * size));
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMalloc((void **)&dc, sizeof(int) * size));
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice));
    gpuErrchk(hipGetLastError());

    int grid = ceil(size * 1.0 / BLOCK_SIZE);
    radix_sort<<<grid, BLOCK_SIZE>>>(da);

    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    hipMemcpy(hc, dc, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(dc);
    hipDeviceReset();

    free(ha);
    free(hc);
 
    high_resolution_clock::time_point end = high_resolution_clock::now();

    free(ha);

    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    for (int i = 0; i < size; i++)
    {
        printf("\t %d\n", hc[i]);
    }

    return time_calc(start, end);
}
