#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>
#include "main_cuda.cuh"

static const int BLOCK_SIZE = 256;

__device__ void swap_device(int *a, int *b)
{
    int t = *a;
    *a = *b;
    *b = t;
}

__device__ int partition_device(int *arr, int low, int high)
{
    int pivot = arr[high];
    int i = (low - 1);

    for (int j = low; j <= high - 1; j++)
    {
        if (arr[j] <= pivot)
        {
            i++;
            swap_device(&arr[i], &arr[j]);
        }
    }
    swap_device(&arr[i + 1], &arr[high]);
    return (i + 1);
}

// Based on CUDA Examples - But Optimized
__global__ void quicksort_device(int *da, int *dc, int left, int right, int size)
{ 
    hipStream_t s1, s2;
    int pi = partition_device(da, left, right);

    int nright = pi - 1;
    int nleft = pi + 1;

    if (left < nright)
    {
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        quicksort_device<<<1, 1, 0, s1>>>(da, dc, left, nright, size);
    }

    if (nleft < right)
    {
        hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
        quicksort_device<<<1, 1, 0, s2>>>(da, dc, nleft, right, size);
    }

    int tid = threadIdx.x;
    if(tid == 0)
        for (int i = 0; i <= size - 1; i++)
            dc[i] = da[i];
}

duration<double> quicksort_gpu_streams(int size)
{
    hipError_t cudaStatus;

    int *ha, *da, *dc;

    ha = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
    }

    high_resolution_clock::time_point start = high_resolution_clock::now();

    cudaStatus = hipMalloc((void **)&da, sizeof(int) * size);
    cudaStatus = hipMalloc((void **)&dc, sizeof(int) * size);

    cudaStatus = hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!  Do you have a CUDA-capable GPU installed?");
        if (abort)
            exit(cudaStatus);
	}

    // hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 16);
    int grid = ceil(size * 1.0 / BLOCK_SIZE);
    quicksort_device<<<grid, BLOCK_SIZE>>>(da, dc, 0, size - 1, size);

    cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed!  Do you have a CUDA-capable GPU installed?");
        if (abort)
            exit(cudaStatus);
    }

    cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed!  Do you have a CUDA-capable GPU installed?");
        if (abort)
            exit(cudaStatus);
    }

    cudaStatus = hipStreamSynchronize(0);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed!  Do you have a CUDA-capable GPU installed?");
        if (abort)
            exit(cudaStatus);
    }

    int *hc = (int *)malloc(sizeof(int) * size);

    cudaStatus = hipMemcpy(hc, dc, sizeof(int) * size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!  Do you have a CUDA-capable GPU installed?");
        if (abort)
            exit(cudaStatus);
	}

    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    for (int i = 0; i < size; i++)
    {
        printf("\t ha[ %d ] => %d\n", i, ha[i]);
        printf("\t hc[ %d ] => %d\n", i, hc[i]);
    }
    
    hipFree(da);
    hipDeviceReset();
    free(ha);

    high_resolution_clock::time_point end = high_resolution_clock::now();
    return time_calc(start, end);
}
