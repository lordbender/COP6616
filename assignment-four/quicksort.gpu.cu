#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>
#include "main_cuda.cuh"

static const int BLOCK_SIZE = 256;

__device__ void swap_device(int *a, int *b)
{
    int t = *a;
    *a = *b;
    *b = t;
}

__device__ int partition_device(int *arr, int low, int high)
{
    int pivot = arr[high];
    int i = (low - 1);

    for (int j = low; j <= high - 1; j++)
    {
        if (arr[j] <= pivot)
        {
            i++;
            swap_device(&arr[i], &arr[j]);
        }
    }
    swap_device(&arr[i + 1], &arr[high]);
    return (i + 1);
}

// Based on CUDA Examples - But Optimized
__global__ void quicksort_device(int *data, int left, int right)
{ 
    hipStream_t s1, s2;
    int pi = partition_device(data, left, right);

    int nright = pi - 1;
    int nleft = pi + 1;

    if (left < nright)
    {
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        quicksort_device<<<1, 32, 0, s1>>>(data, left, nright);
    }

    if (nleft < right)
    {
        hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
        quicksort_device<<<1, 32, 0, s2>>>(data, nleft, right);
    }
}

duration<double> quicksort_gpu_streams(int size)
{
    int* ha = (int *)malloc(sizeof(int) * size);
    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
    }

    high_resolution_clock::time_point start = high_resolution_clock::now();

    int *da;
    gpuErrchk(hipMalloc((void **)&da, sizeof(int) * size));
    gpuErrchk(hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice));

    int grid = ceil(size * 1.0 / BLOCK_SIZE);
    quicksort_device<<<grid, BLOCK_SIZE>>>(da, 0, size - 1);
    gpuErrchk(hipDeviceSynchronize());

    int *hc = (int *)malloc(sizeof(int) * size);
    gpuErrchk(hipMemcpy(hc, da, sizeof(int) * size, hipMemcpyDeviceToHost));

    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    // for (int i = 0; i < size; i++)
    // {
    //     printf("\t hc[ %d ] => %d\n", i, hc[i]);
    // }
    
    gpuErrchk(hipFree(da));
    free(ha);
    free(hc);

    high_resolution_clock::time_point end = high_resolution_clock::now();
    return time_calc(start, end);
}
