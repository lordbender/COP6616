#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>
#include "main_cuda.cuh"

static const int BLOCK_SIZE = 256;

#define MAX_DEPTH       16
#define INSERTION_SORT  32

__global__ void cdp_simple_quicksort(int *data, int left, int right, int depth)
{
    int *lptr = data+left;
    int *rptr = data+right;
    int  pivot = data[(left+right)/2];

    while (lptr <= rptr)
    {
        int lval = *lptr;
        int rval = *rptr;

        while (lval < pivot)
        {
            lptr++;
            lval = *lptr;
        }

        // Move the right pointer as long as the pointed element is larger than the pivot.
        while (rval > pivot)
        {
            rptr--;
            rval = *rptr;
        }

        // If the swap points are valid, do the swap!
        if (lptr <= rptr)
        {
            *lptr++ = rval;
            *rptr-- = lval;
        }
    }

    // Now the recursive part
    int nright = rptr - data;
    int nleft  = lptr - data;

    // Launch a new block to sort the left part.
    if (left < (rptr-data))
    {
        hipStream_t s;
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s >>>(data, left, nright, depth+1);
        hipStreamDestroy(s);
    }

    // Launch a new block to sort the right part.
    if ((lptr-data) < right)
    {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s1 >>>(data, nleft, right, depth+1);
        hipStreamDestroy(s1);
    }
}

// __device__ void swap_device(int *a, int *b)
// {
//     int t = *a;
//     *a = *b;
//     *b = t;
// }

// __device__ int partition_device(int *arr, int low, int high)
// {
//     int pivot = arr[high];
//     int i = (low - 1);

//     for (int j = low; j <= high - 1; j++)
//     {
//         if (arr[j] <= pivot)
//         {
//             i++;
//             swap_device(&arr[i], &arr[j]);
//         }
//     }
//     swap_device(&arr[i + 1], &arr[high]);
//     return (i + 1);
// }

// // Based on CUDA Examples - But Optimized
// __global__ void quicksort_device(int *data, int left, int right)
// { 
//     hipStream_t s1, s2;
//     int pi = partition_device(data, left, right);

//     int nright = pi - 1;
//     int nleft = pi + 1;

//     if (left < nright)
//     {
//         // hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
//         // quicksort_device<<<1, 32, 0, s1>>>(data, left, nright);
//     }

//     if (nleft < right)
//     {
//         // hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
//         // quicksort_device<<<1, 32, 0, s2>>>(data, nleft, right);
//     }
// }

void quicksort_host(int *da, int *hc, int size)
{
    // Call the device.
    int grid = ceil(size * 1.0 / BLOCK_SIZE);
    cdp_simple_quicksort<<<grid, BLOCK_SIZE>>>(da, 0, size - 1, MAX_DEPTH);
    gpuErrchk(hipGetLastError());

    // Ensure the Device is in sync, before we copy the data back!
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipGetLastError());

    // Copy the results back from the device.
    gpuErrchk(hipMemcpy(hc, da, sizeof(int) * size, hipMemcpyDeviceToHost));
    gpuErrchk(hipGetLastError());

    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    for (int i = 0; i < size; i++)
    {
        printf("\t %d\n", hc[i]);
    }
}

duration<double> quicksort_gpu_streams(int size)
{
    int* ha = (int *)malloc(sizeof(int) * size);
    int* hc = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
        hc[i] = 0;
    }

    high_resolution_clock::time_point start = high_resolution_clock::now();

    int *da;
    gpuErrchk(hipMalloc((void **)&da, sizeof(int) * size));
    gpuErrchk(hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice));
    gpuErrchk(hipGetLastError());

    // Kick off the sort!
    quicksort_host(da, hc, size);

    gpuErrchk(hipFree(da));
    free(ha);
    free(hc);
    
    high_resolution_clock::time_point end = high_resolution_clock::now();
    return time_calc(start, end);
}
