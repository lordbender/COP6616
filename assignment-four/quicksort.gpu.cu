
#include <hip/hip_runtime.h>
// #include <stdio.h>
// #include <stdlib.h>
// #include <time.h>

// #include "main_cuda.cuh"

// // A utility function to swap two elements 
// void swap(int* a, int* b) 
// { 
// 	int t = *a; 
// 	*a = *b; 
// 	*b = t; 
// } 

// /* This function takes last element as pivot, places 
// the pivot element at its correct position in sorted 
// 	array, and places all smaller (smaller than pivot) 
// to left of pivot and all greater elements to right 
// of pivot */
// int partition (int arr[], int low, int high) 
// { 
// 	int pivot = arr[high]; // pivot 
// 	int i = (low - 1); // Index of smaller element 

// 	for (int j = low; j <= high- 1; j++) 
// 	{ 
// 		// If current element is smaller than or 
// 		// equal to pivot 
// 		if (arr[j] <= pivot) 
// 		{ 
// 			i++; // increment index of smaller element 
// 			swap(&arr[i], &arr[j]); 
// 		} 
// 	} 
// 	swap(&arr[i + 1], &arr[high]); 
// 	return (i + 1); 
// } 

// void quickSort_gpu(int arr[], int low, int high) 
// { 
// 	if (low < high) 
// 	{ 
// 		int pi = partition(arr, low, high); 

// 		quickSort(arr, low, pi - 1); 
// 		quickSort(arr, pi + 1, high); 
// 	} 
// } 



// // Driver program to test above functions 
// // int main() 
// // { 

// // 	quickSort(arr, 0, n-1); 
// // 	printf("Sorted array: n"); 
// // 	printArray(arr, n); 
// // 	return 0; 
// // } 
