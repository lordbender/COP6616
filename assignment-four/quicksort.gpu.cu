#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>
#include "main_cuda.cuh"

static const int BLOCK_SIZE = 256;

__global__ void quicksort_device(int arr[], int low, int high)
{ 
    hipStream_t s1, s2;

    int pivot_index = low;

	if (low < high) 
	{ 
        int pivot_value = arr[pivot_index];
        int store_index = low;
    
        for (int i = low; i < high; i++)
            if (arr[i] <= pivot_value) {
            
                int temp = arr[i];
                arr[i] = arr[store_index];
                arr[store_index] = temp;

                ++store_index;
            }

            int temp = arr[store_index];
            arr[store_index] = arr[high];
            arr[low] = temp;

        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        quicksort_device<<<1, 64, 0, s1>>>(arr, low, store_index - 1);
        
        hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
        quicksort_device<<<1, 64, 0, s2>>>(arr, store_index + 1, high);
	} 
}

duration<double> quicksort_gpu_streams(int size)
{
    int *ha, *da;

    ha = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
    }

    high_resolution_clock::time_point start = high_resolution_clock::now();

    gpuErrchk(hipMalloc((void **)&da, sizeof(int) * size));
    gpuErrchk(hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice));

    int grid = ceil(size * 1.0 / BLOCK_SIZE);
    quicksort_device<<<grid, BLOCK_SIZE>>>(da, 0, size - 1);
    gpuErrchk(hipDeviceSynchronize());

    int *hc = (int *)malloc(sizeof(int) * size);
    gpuErrchk(hipMemcpy(hc, da, sizeof(int) * size, hipMemcpyDeviceToHost));

    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    for (int i = 0; i < size; i++)
    {
        printf("\t hc[ %d ] => %d\n", i, hc[i]);
    }
    
    gpuErrchk(hipFree(da));
    free(ha);

    high_resolution_clock::time_point end = high_resolution_clock::now();
    return time_calc(start, end);
}
