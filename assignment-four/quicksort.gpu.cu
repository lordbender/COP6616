#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.cuh"

static const int BLOCK_SIZE = 256;

#define swap(A, B)    \
    {                 \
        int temp = A; \
        A = B;        \
        B = temp;     \
    }

typedef struct vars
{
    int l;
    int r;
    int leq;
} vars;

// Portions of this code were based on / modeled after
// https://github.com/khaman1/GPU-QuickSort-Algorithm/blob/master/GPU_quicksort.cu

// Device Portion of Quick Sort
__global__ void gpuPartitionSwap(int *input, int *output, vars *endpts,
                                 int pivot, int l, int r,
                                 int d_leq[],
                                 int d_gt[], int *d_leq_val, int *d_gt_val,
                                 int nBlocks)
{
    __shared__ int bInput[BLOCK_SIZE];
    __syncthreads();
    int idx = l + blockIdx.x * BLOCK_SIZE + threadIdx.x;
    __shared__ int lThisBlock, rThisBlock;
    __shared__ int lOffset, rOffset;

    if (threadIdx.x == 0)
    {
        d_leq[blockIdx.x] = 0;
        d_gt[blockIdx.x] = 0;
        *d_leq_val = 0;
        *d_gt_val = 0;
    }
    __syncthreads();

    if (idx <= (r - 1))
    {
        bInput[threadIdx.x] = input[idx];
        if (bInput[threadIdx.x] <= pivot)
        {
            atomicAdd(&(d_leq[blockIdx.x]), 1);
        }
        else
        {
            atomicAdd(&(d_gt[blockIdx.x]), 1);
        }
    }
    __syncthreads();

    if (threadIdx.x == 0)
    {
        lThisBlock = d_leq[blockIdx.x];
        lOffset = l + atomicAdd(d_leq_val, lThisBlock);
    }
    if (threadIdx.x == 1)
    {
        rThisBlock = d_gt[blockIdx.x];
        rOffset = r - atomicAdd(d_gt_val, rThisBlock);
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {

        int m = 0;
        int n = 0;
        for (int j = 0; j < BLOCK_SIZE; j++)
        {
            int chk = l + blockIdx.x * BLOCK_SIZE + j;
            if (chk <= (r - 1))
            {
                if (bInput[j] <= pivot)
                {
                    output[lOffset + m] = bInput[j];
                    ++m;
                }
                else
                {
                    output[rOffset - n] = bInput[j];
                    ++n;
                }
            }
        }
    }

    __syncthreads();

    if ((blockIdx.x == 0) && (threadIdx.x == 0))
    {
        int pOffset = l;
        for (int k = 0; k < nBlocks; k++)
            pOffset += d_leq[k];

        output[pOffset] = pivot;
        endpts->l = (pOffset - 1);
        endpts->r = (pOffset + 1);
    }

    return;
}

// Host Portion of Quick Sort

void quicksort(int *ls, int l, int r, int length)
{
    if ((r - l) >= 1)
    {
        int pivot = ls[r];

        int numBlocks = (r - l) / BLOCK_SIZE;
        if ((numBlocks * BLOCK_SIZE) < (r - l))
            numBlocks++;

        int *d_ls;
        int *d_ls2;
        vars endpts;
        endpts.l = l;
        endpts.r = r;

        vars *d_endpts;
        int *d_leq, *d_gt, *d_leq_val, *d_gt_val;
        int size = sizeof(int);
		hipMalloc(&(d_ls), size * length);
		gpuErrchk(hipGetLastError());

		hipMalloc(&(d_ls2), size * length);
		gpuErrchk(hipGetLastError());

		hipMalloc(&(d_endpts), sizeof(vars));
		gpuErrchk(hipGetLastError());

		hipMalloc(&(d_leq), 4 * numBlocks);
		gpuErrchk(hipGetLastError());

		hipMalloc(&(d_gt), 4 * numBlocks);
		gpuErrchk(hipGetLastError());

		hipMalloc(&d_leq_val, 4);
		gpuErrchk(hipGetLastError());

		hipMalloc(&d_gt_val, 4);
		gpuErrchk(hipGetLastError());
				
		hipMemcpy(d_ls, ls, size * length, hipMemcpyHostToDevice);
		gpuErrchk(hipGetLastError());

        hipMemcpy(d_ls2, ls, size * length, hipMemcpyHostToDevice);
		gpuErrchk(hipGetLastError());

        gpuPartitionSwap<<<numBlocks, BLOCK_SIZE>>>(d_ls, d_ls2, d_endpts, pivot, l, r, d_leq, d_gt, d_leq_val, d_gt_val, numBlocks);
		gpuErrchk(hipGetLastError());

		hipMemcpy(ls, d_ls2, size * length, hipMemcpyDeviceToHost);
		gpuErrchk(hipGetLastError());

		hipMemcpy(&(endpts), d_endpts, sizeof(vars), hipMemcpyDeviceToHost);
		gpuErrchk(hipGetLastError());
		
        hipDeviceSynchronize();

        hipFree(d_ls);
        hipFree(d_ls2);
        hipFree(d_endpts);
        hipFree(d_leq);
        hipFree(d_gt);

        if (endpts.l >= l)
			quicksort(ls, l, endpts.l, length);
        if (endpts.r <= r)
			quicksort(ls, endpts.r, r, length);
    }

    return;
}

double quicksort_gpu(int size)
{
    int *ha = (int *)malloc(sizeof(int) * size);

    clock_t start = clock();
    quicksort(ha, 0, size - 1, size);
    clock_t end = clock();

    return time_calc(start, end);
}