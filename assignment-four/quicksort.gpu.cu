#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>
#include "main_cuda.cuh"

static const int BLOCK_SIZE = 256;

__device__ void swap_device(int *a, int *b)
{
    int t = *a;
    *a = *b;
    *b = t;
}

__device__ int partition_device(int *arr, int low, int high)
{
    int pivot = arr[high];
    int i = (low - 1);

    for (int j = low; j <= high - 1; j++)
    {
        if (arr[j] <= pivot)
        {
            i++;
            swap_device(&arr[i], &arr[j]);
        }
    }
    swap_device(&arr[i + 1], &arr[high]);
    return (i + 1);
}

// Based on CUDA Examples - But Optimized
__global__ void quicksort_device(int *data, int left, int right)
{
    int pi = partition_device(data, left, right);

    int nright = pi - 1;
    int nleft = pi + 1;

    if (left < nright)
    {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        quicksort_device<<<1, 64, 0, s1>>>(data, left, nright);
        hipStreamDestroy(s1);
    }

    if (nleft < right)
    {
        hipStream_t s2;
        hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
        quicksort_device<<<1, 64, 0, s2>>>(data, nleft, right);
        hipStreamDestroy(s2);
    }

    return;
}

duration<double> quicksort_gpu_streams(int size)
{
    hipError_t cudaStatus;

    int *ha, *da, *hc;

    ha = (int *)malloc(sizeof(int) * size);
    hc = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
        hc[i] = 0;
    }

    high_resolution_clock::time_point start = high_resolution_clock::now();

    cudaStatus = hipMalloc((void **)&da, sizeof(int) * size);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!  Do you have a CUDA-capable GPU installed?");
        if (abort)
            exit(cudaStatus);
	}

    cudaStatus = hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!  Do you have a CUDA-capable GPU installed?");
        if (abort)
            exit(cudaStatus);
	}

    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 16);
    int grid = ceil(size * 1.0 / BLOCK_SIZE);
    quicksort_device<<<grid, BLOCK_SIZE>>>(da, 0, size - 1);
    hipDeviceSynchronize();

    cudaStatus = hipMemcpy(hc, da, sizeof(int) * size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!  Do you have a CUDA-capable GPU installed?");
        if (abort)
            exit(cudaStatus);
	}

	hipFree(da);
    hipDeviceReset();
    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    for (int i = 0; i < size; i++)
    {
        printf("\t %d\n", hc[i]);
    }

    free(ha);

    high_resolution_clock::time_point end = high_resolution_clock::now();
    return time_calc(start, end);
}