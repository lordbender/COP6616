#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.cuh"

static const int BLOCK_SIZE = 256;

// Device code
__global__ void vecSquare(int* a, int* c, int n)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
        c[id] = a[id] * a[id];
}

double quickSort_gpu(int size) {
	int *ha, *hc, *da, *dc;

    ha = (int*)malloc(sizeof(int) * size);
	hc = (int*)malloc(sizeof(int) * size);
	
	for (int i = 0; i < size; i++)
	{
		ha[i] = rand();
		hc[i] = 0;
	}

	clock_t start = clock();

	gpuErrchk(hipMalloc((void**) &da, sizeof(int) * size));
	gpuErrchk(hipGetLastError());

	gpuErrchk(hipMalloc((void**) &dc, sizeof(int) * size));
	gpuErrchk(hipGetLastError());

	gpuErrchk(hipMemcpy(da, ha,  sizeof(int) * size, hipMemcpyHostToDevice));
	gpuErrchk(hipGetLastError());

	int grid = ceil(size * 1.0 / BLOCK_SIZE);
	vecSquare<<<grid, BLOCK_SIZE>>>(da, dc, size);
	hipDeviceSynchronize();
	gpuErrchk(hipGetLastError());

	hipMemcpy(hc, dc, sizeof(int) * size, hipMemcpyDeviceToHost);

	hipFree(da);
	hipFree(dc);
	hipDeviceReset();

	free(ha);
	free(hc);
	
	clock_t end = clock();

	// Testing that sort is working, keep commented out on large values of N (say N > 1000)
	// for (int i = 0; i < size; i++) {
    // 	printf("\t %d\n", hc[i]);
	// }

	return time_calc(start, end);
}

// // A utility function to swap two elements 
// void swap(int* a, int* b) 
// { 
// 	int t = *a; 
// 	*a = *b; 
// 	*b = t; 
// } 

// /* This function takes last element as pivot, places 
// the pivot element at its correct position in sorted 
// 	array, and places all smaller (smaller than pivot) 
// to left of pivot and all greater elements to right 
// of pivot */
// int partition (int arr[], int low, int high) 
// { 
// 	int pivot = arr[high]; // pivot 
// 	int i = (low - 1); // Index of smaller element 

// 	for (int j = low; j <= high- 1; j++) 
// 	{ 
// 		// If current element is smaller than or 
// 		// equal to pivot 
// 		if (arr[j] <= pivot) 
// 		{ 
// 			i++; // increment index of smaller element 
// 			swap(&arr[i], &arr[j]); 
// 		} 
// 	} 
// 	swap(&arr[i + 1], &arr[high]); 
// 	return (i + 1); 
// } 

// void quickSort_gpu(int arr[], int low, int high) 
// { 
// 	if (low < high) 
// 	{ 
// 		int pi = partition(arr, low, high); 

// 		quickSort(arr, low, pi - 1); 
// 		quickSort(arr, pi + 1, high); 
// 	} 
// } 



// // Driver program to test above functions 
// // int main() 
// // { 

// // 	quickSort(arr, 0, n-1); 
// // 	printf("Sorted array: n"); 
// // 	printArray(arr, n); 
// // 	return 0; 
// // } 
