#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.cuh"

static const int BLOCK_SIZE = 256;

__device__ void swap_device(int* a, int* b) 
{ 
	int t = *a; 
	*a = *b; 
	*b = t; 
} 

__device__ int partition_device (int arr[], int low, int high) 
{ 
	int pivot = arr[high];
	int i = (low - 1);

	for (int j = low; j <= high- 1; j++) 
	{ 
		if (arr[j] <= pivot) 
		{ 
			i++;
			swap_device(&arr[i], &arr[j]); 
		} 
	} 
	swap_device(&arr[i + 1], &arr[high]); 
	return (i + 1); 
} 

__global__ void quicksort_device(int *arr, int low, int high, int grid, int block)
{
	if (low < high) 
	{ 
		int pi = partition_device(arr, low, high); 

		quicksort_device<<<grid, block>>>(arr, low, pi - 1, grid, block); 
		quicksort_device<<<grid, block>>>(arr, pi + 1, high, grid, block); 
	} 

	// if (left < right){
	// 	hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
	// 	quicksort_device<<< ..., s1 >>> (data, left, right);
	// }

	// if (nleft < right){
	// 	hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
	// 	quicksort_device<<< ..., s2 >>> (data, nleft, right);
	// }

    // return;
}

double quicksort_gpu(int size)
{
	int *ha, *da;

    ha = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
    }

    clock_t start = clock();

    gpuErrchk(hipMalloc((void **)&da, sizeof(int) * size));
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice));
    gpuErrchk(hipGetLastError());

	int grid = ceil(size * 1.0 / BLOCK_SIZE);
	
    quicksort_device<<<grid, BLOCK_SIZE>>>(da, 0, size - 1, grid, BLOCK_SIZE);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    hipMemcpy(ha, da, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(da);
    hipDeviceReset();

    free(ha);

    clock_t end = clock();
    return time_calc(start, end);
}