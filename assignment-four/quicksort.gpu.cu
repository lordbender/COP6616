#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>
#include "main_cuda.cuh"

#define MAX_DEPTH       16

static const int BLOCK_SIZE = 256;

__device__ void selection_sort(int *data, int left, int right)
{
    for (int i = left ; i <= right ; ++i)
    {
        int min_val = data[i];
        int min_idx = i;

        // Find the smallest value in the range [left, right].
        for (int j = i+1 ; j <= right ; ++j)
        {
            int val_j = data[j];

            if (val_j < min_val)
            {
                min_idx = j;
                min_val = val_j;
            }
        }

        // Swap the values.
        if (i != min_idx)
        {
            data[min_idx] = data[i];
            data[i] = min_val;
        }
    }
}

__device__ void swap_device(int *a, int *b)
{
    int t = *a;
    *a = *b;
    *b = t;
}

__device__ int partition_device(int *arr, int low, int high)
{
    int pivot = arr[high];
    int i = (low - 1);

    for (int j = low; j <= high - 1; j++)
    {
        if (arr[j] <= pivot)
        {
            i++;
            swap_device(&arr[i], &arr[j]);
        }
    }
    swap_device(&arr[i + 1], &arr[high]);
    return (i + 1);
}

// Based on CUDA Examples - But Optimized
__global__ void quicksort_device(int *data, int left, int right, int depth)
{
    // If we're too deep or there are few elements left, we use an insertion sort...
    if (depth >= MAX_DEPTH || right-left <= 32)
    {
        selection_sort(data, left, right);
        return;
    }

    int *lptr = data+left;
    int *rptr = data+right;
    int  pivot = data[(left+right)/2];

    // Do the partitioning.
    while (lptr <= rptr)
    {
        // Find the next left- and right-hand values to swap
        int lval = *lptr;
        int rval = *rptr;

        // Move the left pointer as long as the pointed element is smaller than the pivot.
        while (lval < pivot)
        {
            lptr++;
            lval = *lptr;
        }

        // Move the right pointer as long as the pointed element is larger than the pivot.
        while (rval > pivot)
        {
            rptr--;
            rval = *rptr;
        }

        // If the swap points are valid, do the swap!
        if (lptr <= rptr)
        {
            *lptr++ = rval;
            *rptr-- = lval;
        }
    }

    // Now the recursive part
    int nright = rptr - data;
    int nleft  = lptr - data;

    // Launch a new block to sort the left part.
    if (left < (rptr-data))
    {
        hipStream_t s;
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        quicksort_device<<< 1, 1, 0, s >>>(data, left, nright, depth+1);
        hipStreamDestroy(s);
    }

    // Launch a new block to sort the right part.
    if ((lptr-data) < right)
    {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        quicksort_device<<< 1, 1, 0, s1 >>>(data, nleft, right, depth+1);
        hipStreamDestroy(s1);
    }
}

duration<double> quicksort_gpu_streams(int size)
{
    hipError_t cudaStatus;

    int *ha, *da, *hc;

    ha = (int *)malloc(sizeof(int) * size);
    hc = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
        hc[i] = 0;
    }

    high_resolution_clock::time_point start = high_resolution_clock::now();

    cudaStatus = hipMalloc((void **)&da, sizeof(int) * size);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!  Do you have a CUDA-capable GPU installed?");
        if (abort)
            exit(cudaStatus);
	}

    cudaStatus = hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!  Do you have a CUDA-capable GPU installed?");
        if (abort)
            exit(cudaStatus);
	}

    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH);
    int grid = ceil(size * 1.0 / BLOCK_SIZE);
    quicksort_device<<<grid, BLOCK_SIZE>>>(da, 0, size - 1, 0);
    hipDeviceSynchronize();

    cudaStatus = hipMemcpy(hc, da, sizeof(int) * size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!  Do you have a CUDA-capable GPU installed?");
        if (abort)
            exit(cudaStatus);
	}

	hipFree(da);
    hipDeviceReset();
    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    for (int i = 0; i < size; i++)
    {
        printf("\t %d\n", hc[i]);
    }

    free(ha);

    high_resolution_clock::time_point end = high_resolution_clock::now();
    return time_calc(start, end);
}