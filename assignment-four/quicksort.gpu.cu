#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.cuh"

static const int BLOCK_SIZE = 256;

__device__ void swap_device(int *a, int* b) 
{ 
	int t = *a; 
	*a = *b; 
	*b = t; 
} 

__device__ int partition_device (int *arr, int low, int high) 
{ 
	int pivot = arr[high];
	int i = (low - 1);

	for (int j = low; j <= high- 1; j++) 
	{ 
		if (arr[j] <= pivot) 
		{ 
			i++;
			swap_device(&arr[i], &arr[j]); 
		} 
	} 
	swap_device(&arr[i + 1], &arr[high]); 
	return (i + 1); 
} 

__global__ void quicksort_device(int *arr, int low, int high)
{
	hipStream_t s1, s2;

	if (low < high){
		int pi = partition_device(arr, low, high); 

		hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);

		quicksort_device<<<1, 64, 0, s1>>>(arr, low, pi - 1);
		quicksort_device<<<1, 64, 0, s2>>>(arr, pi + 1, high);
	}
	return;
}

double quicksort_gpu(int size)
{
	int *ha, *da;

    ha = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
    }

    clock_t start = clock();

    gpuErrchk(hipMalloc((void **)&da, sizeof(int) * size));
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice));
    gpuErrchk(hipGetLastError());

	int grid = ceil(size * 1.0 / BLOCK_SIZE);
	
    quicksort_device<<<grid, BLOCK_SIZE>>>(da, 0, size - 1);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    hipMemcpy(ha, da, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(da);
    hipDeviceReset();

    free(ha);

    clock_t end = clock();
    return time_calc(start, end);
}