#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.cuh"

static const int BLOCK_SIZE = 256;

__global__ void swap(int* a, int* b) 
{ 
	int t = *a; 
	*a = *b; 
	*b = t; 
} 

__global__ int partition (int arr[], int low, int high) 
{ 
	int pivot = arr[high];
	int i = (low - 1);

	for (int j = low; j <= high- 1; j++) 
	{ 
		if (arr[j] <= pivot) 
		{ 
			i++;
			swap(&arr[i], &arr[j]); 
		} 
	} 
	swap(&arr[i + 1], &arr[high]); 
	return (i + 1); 
} 

__global__ void quicksort_device(int *data, int left, int right)
{
	int nleft, nright;
	hipStream_t s1, s2;	

	partition(data, nleft, nright);

	if (left < right){
		hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		quicksort_device<<< ..., s1 >>> (data, left, right);
	}

	if (nleft < right){
		hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
		quicksort_device<<< ..., s2 >>> (data, nleft, right);
	}

    return;
}

double quicksort_gpu(int size)
{
	int *ha, *da;

    ha = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
    }

    clock_t start = clock();

    gpuErrchk(hipMalloc((void **)&da, sizeof(int) * size));
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice));
    gpuErrchk(hipGetLastError());

	int grid = ceil(size * 1.0 / BLOCK_SIZE);
	
    quicksort_device<<<grid, BLOCK_SIZE>>>(da, 0, size - 1);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    hipMemcpy(ha, da, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(da);
    hipDeviceReset();

    free(ha);

    clock_t end = clock();
    return time_calc(start, end);
}