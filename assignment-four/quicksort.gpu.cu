#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>
#include "main_cuda.cuh"

static const int BLOCK_SIZE = 256;

__device__ void swap_device(unsigned int *a, unsigned int *b)
{
    unsigned int t = *a;
    *a = *b;
    *b = t;
}

__device__ unsigned int partition_device(unsigned int *arr, int low, int high)
{
    unsigned int pivot = arr[high];
    int i = (low - 1);

    for (int j = low; j <= high - 1; j++)
    {
        if (arr[j] <= pivot)
        {
            i++;
            swap_device(&arr[i], &arr[j]);
        }
    }
    swap_device(&arr[i + 1], &arr[high]);
    return (i + 1);
}

// Based on CUDA Examples - But Optimized
__global__ void quicksort_device(unsigned int *data, int left, int right)
{
    int pi = partition_device(data, left, right);

    int nright = pi - 1;
    int nleft = pi + 1;

    if (left < nright)
    {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        quicksort_device<<<1, 1, 0, s1>>>(data, left, nright);
        hipStreamDestroy(s1);
    }

    if (nleft < right)
    {
        hipStream_t s2;
        hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
        quicksort_device<<<1, 1, 0, s2>>>(data, nleft, right);
        hipStreamDestroy(s2);
    }
}

duration<double> quicksort_gpu_streams(int size)
{
    hipError_t cudaStatus;

    unsigned int *ha, *da;

    ha = (unsigned int *)malloc(sizeof(unsigned int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand() % size;
    }

    high_resolution_clock::time_point start = high_resolution_clock::now();

    cudaStatus = hipMalloc((void **)&da, sizeof(unsigned int) * size);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!  Do you have a CUDA-capable GPU installed?");
        if (abort)
            exit(cudaStatus);
	}

    cudaStatus = hipMemcpy(da, ha, sizeof(unsigned int) * size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!  Do you have a CUDA-capable GPU installed?");
        if (abort)
            exit(cudaStatus);
	}

    // hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 16);
    int grid = ceil(size * 1.0 / BLOCK_SIZE);
    quicksort_device<<<grid, BLOCK_SIZE>>>(da, 0, size - 1);
    cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed!  Do you have a CUDA-capable GPU installed?");
        if (abort)
            exit(cudaStatus);
    }

    unsigned int *hc = (unsigned int *)malloc(sizeof(unsigned int) * size);
    cudaStatus = hipMemcpy(hc, da, sizeof(unsigned int) * size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!  Do you have a CUDA-capable GPU installed?");
        if (abort)
            exit(cudaStatus);
	}

    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    for (int i = 0; i < size; i++)
    {
        printf("\t hc[ %d ] => %d\n", i, hc[i]);
    }
    
    hipFree(da);
    hipDeviceReset();
    free(ha);

    high_resolution_clock::time_point end = high_resolution_clock::now();
    return time_calc(start, end);
}
