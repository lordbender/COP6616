#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>
#include "main_cuda.cuh"

static const int BLOCK_SIZE = 256;

__device__ void swap_device(int array[], int left, int right)
{
	int temp;
	temp = array[left];
	array[left] = array[right];
	array[right] = temp;
}

__device__ int partition_device(int array[], int left, int right, int pivot_index)
{
	int pivot_value = array[pivot_index];
	int store_index = left;
	int i;

	swap_device(array, pivot_index, right);
	for (i = left; i < right; i++)
		if (array[i] <= pivot_value) {
			swap_device(array, i, store_index);
			++store_index;
		}
	swap_device(array, store_index, right);
	return store_index;
}
// Based on CUDA Examples - But Optimized
__global__ void quicksort_device(int arr[], int low, int high)
{ 
    hipStream_t s1, s2;

    int pivot_index = low;

	if (low < high) 
	{ 
		int pi = partition_device(arr, low, high, pivot_index); 

        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        quicksort_device<<<1, 64, 0, s1>>>(arr, low, pi - 1);
        
        hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
        quicksort_device<<<1, 64, 0, s2>>>(arr, pi + 1, high);
	} 
}

duration<double> quicksort_gpu_streams(int size)
{
    int *ha, *da;

    ha = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
    }

    high_resolution_clock::time_point start = high_resolution_clock::now();

    gpuErrchk(hipMalloc((void **)&da, sizeof(int) * size));
    gpuErrchk(hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice));

    int grid = ceil(size * 1.0 / BLOCK_SIZE);
    quicksort_device<<<grid, BLOCK_SIZE>>>(da, 0, size - 1);
    gpuErrchk(hipDeviceSynchronize());

    int *hc = (int *)malloc(sizeof(int) * size);
    gpuErrchk(hipMemcpy(hc, da, sizeof(int) * size, hipMemcpyDeviceToHost));

    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    for (int i = 0; i < size; i++)
    {
        printf("\t hc[ %d ] => %d\n", i, hc[i]);
    }
    
    gpuErrchk(hipFree(da));
    free(ha);

    high_resolution_clock::time_point end = high_resolution_clock::now();
    return time_calc(start, end);
}
