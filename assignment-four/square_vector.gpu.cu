
#include <hip/hip_runtime.h>
// // This is a good place to start looking to 
// // grasp teh basics.

// #include <stdio.h>
// #include <stdlib.h>
// #include <time.h>

// #include "main_cuda.cuh"

// static const int BLOCK_SIZE = 256;

// // Device Portion of Quick Sort
// __global__ void vecSquare(int *a, int *c, int n)
// {
//     int id = blockIdx.x * blockDim.x + threadIdx.x;
//     if (id < n)
//         c[id] = a[id] * a[id];
// }

// // Device Portion of Quick Sort
// double square_vector_gpu(int size)
// {
//     int *ha, *hc, *da, *dc;

//     ha = (int *)malloc(sizeof(int) * size);
//     hc = (int *)malloc(sizeof(int) * size);

//     for (int i = 0; i < size; i++)
//     {
//         ha[i] = rand();
//         hc[i] = 0;
//     }

//     clock_t start = clock();

//     gpuErrchk(cudaMalloc((void **)&da, sizeof(int) * size));
//     gpuErrchk(cudaGetLastError());

//     gpuErrchk(cudaMalloc((void **)&dc, sizeof(int) * size));
//     gpuErrchk(cudaGetLastError());

//     gpuErrchk(cudaMemcpy(da, ha, sizeof(int) * size, cudaMemcpyHostToDevice));
//     gpuErrchk(cudaGetLastError());

//     int grid = ceil(size * 1.0 / BLOCK_SIZE);
//     vecSquare<<<grid, BLOCK_SIZE>>>(da, dc, size);
//     cudaDeviceSynchronize();
//     gpuErrchk(cudaGetLastError());

//     cudaMemcpy(hc, dc, sizeof(int) * size, cudaMemcpyDeviceToHost);

//     cudaFree(da);
//     cudaFree(dc);
//     cudaDeviceReset();

//     free(ha);
//     free(hc);

//     clock_t end = clock();

//     // Testing that sort is working, keep commented out on large values of N (say N > 1000)
//     // for (int i = 0; i < size; i++) {
//     // 	printf("\t %d\n", hc[i]);
//     // }

//     return time_calc(start, end);
// }
