#include "hip/hip_runtime.h"
// This is a good place to start looking to 
// grasp teh basics.

#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>

#include "main_cuda.cuh"

using namespace std::chrono;

static const int BLOCK_SIZE = 256;

// Device Portion of Quick Sort
__global__ void vecSquare(int *a, int *c, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
        c[id] = a[id] * a[id];
}

// Device Portion of Quick Sort
duration<double> square_vector_gpu(int size)
{
    int *ha, *hc, *da, *dc;

    ha = (int *)malloc(sizeof(int) * size);
    hc = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
        hc[i] = 0;
    }

    high_resolution_clock::time_point start = high_resolution_clock::now();

    gpuErrchk(hipMalloc((void **)&da, sizeof(int) * size));
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMalloc((void **)&dc, sizeof(int) * size));
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice));
    gpuErrchk(hipGetLastError());

    int grid = ceil(size * 1.0 / BLOCK_SIZE);
    vecSquare<<<grid, BLOCK_SIZE>>>(da, dc, size);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    hipMemcpy(hc, dc, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(dc);
    hipDeviceReset();

    free(ha);
    free(hc);

    high_resolution_clock::time_point end = high_resolution_clock::now();

    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    // for (int i = 0; i < size; i++) {
    // 	printf("\t %d\n", hc[i]);
    // }

    return time_calc(start, end);
}
