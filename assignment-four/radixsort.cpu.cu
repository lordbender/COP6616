#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>
#include <vector>

#include "main_cuda.cuh"

using namespace std::chrono;

int getMax(int *arr, int n) 
{ 
    int mx = arr[0]; 
    for (int i = 1; i < n; i++) 
        if (arr[i] > mx) 
            mx = arr[i]; 
    return mx; 
} 
  
// A function to do counting sort of arr[] according to 
// the digit represented by exp. 
void countSort(int *arr, int n, int exp) 
{ 
    int *output = (int *)malloc(sizeof(int) * n); 
    int i, count[10] = {0}; 
  
    for (i = 0; i < n; i++) 
        count[ (arr[i]/exp)%10 ]++; 

    for (i = 1; i < 10; i++) 
        count[i] += count[i - 1]; 
  
    // Build the output array 
    for (i = n - 1; i >= 0; i--) 
    { 
        output[count[ (arr[i]/exp)%10 ] - 1] = arr[i]; 
        count[ (arr[i]/exp)%10 ]--; 
    } 

    for (i = 0; i < n; i++) 
        arr[i] = output[i]; 
} 
  
void radixsort(int *arr, int n) 
{ 
    int m = getMax(arr, n); 
  
    for (int exp = 1; m/exp > 0; exp *= 10) 
        countSort(arr, n, exp); 
}


duration<double> radixsort_cpu(int size)
{   
    int *a = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        a[i] = rand();
    }
 
  	high_resolution_clock::time_point start = high_resolution_clock::now();
    radixsort(a, size); 
    high_resolution_clock::time_point end = high_resolution_clock::now();
	
    // Testing that sort is working, keep commented out on large values of N (say N > 1000)
    // for (int i = 0; i < size; i++)
    // {
    //     printf("\t %d\n", a[i]);
    // }

    return time_calc(start, end);
}