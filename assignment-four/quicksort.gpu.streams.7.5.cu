#include "hip/hip_runtime.h"
// http://on-demand.gputechconf.com/gtc/2014/presentations/S4158-cuda-streams-best-practices-common-pitfalls.pdf
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>
#include <vector>

#include "main_cuda.cuh"

using namespace std::chrono;

static const int BLOCK_SIZE = 256;

__global__ void quicksort_device(int *a) {
    // do the work!!!

    return;
}

void quicksort_host(int *da, int left, int right, int size)
{
	int pivot_index = left;
    int pivot_new_index = partition(da, left, right, pivot_index);
    
    // int new_right = pivot_new_index - 1;
    // int new_left = pivot_new_index + 1;

    // Create a thread pool on the GPU
    // Use Streams to Parallelize on the GPU tobreduce copy footprint.
    hipStream_t s1, s2;
    hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);

    // split up the work so we can thread it!!!
    int *db = (int*)malloc(sizeof(int) * 1);
    int *dc = (int*)malloc(sizeof(int) * 1);

    // partition based on the pivot;
    int grid = ceil(size * 1.0 / BLOCK_SIZE);
    quicksort_device<<<grid, BLOCK_SIZE, 0, s1>>>(db);
    quicksort_device<<<grid, BLOCK_SIZE, 0, s2>>>(dc);

	hipStreamSynchronize(0);
    gpuErrchk(hipGetLastError());

    // Clean up the thread pool.
    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
}

duration<double> quicksort_gpu_streams(int size)
{
    int *ha, *da;

    ha = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
    }

    high_resolution_clock::time_point start = high_resolution_clock::now();

    gpuErrchk(hipMalloc((void **)&da, sizeof(int) * size));
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice));
    gpuErrchk(hipGetLastError());

    quicksort_host(da, 0, size - 1, size);

    hipMemcpy(ha, da, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(da);
    hipDeviceReset();

    free(ha);

    high_resolution_clock::time_point end = high_resolution_clock::now();

    return time_calc(start, end);
}