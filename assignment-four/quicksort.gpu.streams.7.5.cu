#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>

#include "main_cuda.cuh"

using namespace std::chrono;

static const int BLOCK_SIZE = 256;

// __device__ void swap_device(int *a, int *b)
// {
//     int t = *a;
//     *a = *b;
//     *b = t;
// }

// __device__ int partition_device(int *arr, int low, int high)
// {
//     int pivot = arr[high];
//     int i = (low - 1);

//     for (int j = low; j <= high - 1; j++)
//     {
//         if (arr[j] <= pivot)
//         {
//             i++;
//             swap_device(&arr[i], &arr[j]);
//         }
//     }
//     swap_device(&arr[i + 1], &arr[high]);
//     return (i + 1);
// }

__global__ void quicksort_device(int *data, int left, int right)
{

}

duration<double> quicksort_gpu_streams(int size)
{
    int *ha, *da;

    ha = (int *)malloc(sizeof(int) * size);

    for (int i = 0; i < size; i++)
    {
        ha[i] = rand();
    }

    high_resolution_clock::time_point start = high_resolution_clock::now();

    gpuErrchk(hipMalloc((void **)&da, sizeof(int) * size));
    gpuErrchk(hipGetLastError());

    gpuErrchk(hipMemcpy(da, ha, sizeof(int) * size, hipMemcpyHostToDevice));
    gpuErrchk(hipGetLastError());

    int grid = ceil(size * 1.0 / BLOCK_SIZE);

    quicksort_device<<<grid, BLOCK_SIZE>>>(da, 0, size - 1);

	hipStreamSynchronize(0);
    gpuErrchk(hipGetLastError());

    hipMemcpy(ha, da, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(da);
    hipDeviceReset();

    free(ha);

    high_resolution_clock::time_point end = high_resolution_clock::now();

    return time_calc(start, end);
}