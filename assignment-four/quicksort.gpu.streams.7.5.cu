
#include <hip/hip_runtime.h>
// // http://on-demand.gputechconf.com/gtc/2014/presentations/S4158-cuda-streams-best-practices-common-pitfalls.pdf
// #include <stdio.h>
// #include <stdlib.h>
// #include <ctime>
// #include <ratio>
// #include <chrono>
// #include <vector>

// #include "main_cuda.cuh"

// using namespace std::chrono;

// static const int BLOCK_SIZE = 256;

// __global__ void quicksort_device(int *array, int left, int right) {
//     // do the work!!!

//     return;
// }

// void quicksort_host(int *ha, int left, int right, int size, int depth)
// {
//     if (right > left)
//     {
//         int pivotIndex = left + (right - left)/2;
//         pivotIndex = partition(ha, left, right, pivotIndex);

//         int new_right = pivotIndex - 1;
//         int new_left = pivotIndex + 1;
        
//         if (depth-- > 0)
//         {
//             int *da;

//             gpuErrchk(cudaMalloc((void **)&da, sizeof(int) * size));
//             gpuErrchk(cudaGetLastError());
        
//             gpuErrchk(cudaMemcpy(da, ha, sizeof(int) * size, cudaMemcpyHostToDevice));
//             gpuErrchk(cudaGetLastError());

//             int grid = ceil(size * 1.0 / BLOCK_SIZE);

//             quicksort_device<<<grid, BLOCK_SIZE>>>(da, left, new_right);
//             quicksort_device<<<grid, BLOCK_SIZE>>>(da, new_left, right);

//             cudaDeviceSynchronize();
//             gpuErrchk(cudaGetLastError());

//             cudaMemcpy(ha, da, sizeof(int) * size, cudaMemcpyDeviceToHost);

//             cudaFree(da);
//             cudaDeviceReset();
//         }
//         else
//         {
//             quicksort(ha, left, new_right);
//             quicksort(ha, new_left, right);
//         }
//     }
// }

// duration<double> quicksort_gpu_streams(int size)
// {
//     int *ha  = (int *)malloc(sizeof(int) * size);

//     for (int i = 0; i < size; i++)
//     {
//         ha[i] = rand();
//     }

//     high_resolution_clock::time_point start = high_resolution_clock::now();
//     quicksort_host(ha, 0, size - 1, size, 5);    
//     high_resolution_clock::time_point end = high_resolution_clock::now();

//     free(ha);

//     // Testing that sort is working, keep commented out on large values of N (say N > 1000)
//     // for (int i = 0; i < size; i++)
//     // {
//     //     printf("\t %d\n", ha[i]);
//     // }

//     return time_calc(start, end);
// }

// // void quicksort_host(int *da, int left, int right, int size)
// // {
// // 	// int pivot_index = left;
// //     // int pivot_new_index = partition(da, left, right, pivot_index);
    
// //     // int new_right = pivot_new_index - 1;
// //     // int new_left = pivot_new_index + 1;

// //     // Create a thread pool on the GPU
// //     // Use Streams to Parallelize on the GPU tobreduce copy footprint.
// //     cudaStream_t s1, s2;
// //     cudaStreamCreateWithFlags(&s1, cudaStreamNonBlocking);
// //     cudaStreamCreateWithFlags(&s2, cudaStreamNonBlocking);

// //     // split up the work so we can thread it!!!
// //     int *db = (int*)malloc(sizeof(int) * 1);
// //     int *dc = (int*)malloc(sizeof(int) * 1);

// //     // partition based on the pivot;
// //     int grid = ceil(size * 1.0 / BLOCK_SIZE);
// //     quicksort_device<<<grid, BLOCK_SIZE, 0, s1>>>(db);
// //     quicksort_device<<<grid, BLOCK_SIZE, 0, s2>>>(dc);

// // 	cudaStreamSynchronize(0);
// //     gpuErrchk(cudaGetLastError());

// //     // Clean up the thread pool.
// //     cudaStreamDestroy(s1);
// //     cudaStreamDestroy(s2);
// // }