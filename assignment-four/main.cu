#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.cuh"
 
// See: https://devblogs.nvidia.com/gpu-pro-tip-cuda-7-streams-simplify-concurrency/
int main(int argc, char *argv[])
{
  	srand(time(0));

 	int size = atoi(argv[1]);

    /* ------------------------ Begin CPU Sequential Benchmarking ------------------------ */
	
	double cpu_runtime = quicksort_cpu(size);
	printf("\n\tCPU O(n*log(n)) Sequential: Completed %d numbers in %f seconds!!!\n", size, cpu_runtime);

	/* ------------------------ End CPU Sequential Benchmarking ------------------------ */


	/* ------------------------ Begin CPU Parallel Benchmarking ------------------------ */

	double cpu_threads_runtime = quicksort_cpu_threads(size);
	printf("\tCPU O(n*log(n)) Threaded: Completed %d numbers in %f seconds!!!\n\n", size, cpu_threads_runtime);
	
	/* ------------------------ END GPU Parallel Benchmarking ------------------------ */

	
	/* ------------------------ Begin GPU Parallel Benchmarking ------------------------ */

	double gpu_streams_runtime = quicksort_gpu_streams(size);
	printf("\tGPU O(n*log(n)) Streamed: Completed %d numbers in %f seconds!!!\n\n", size, gpu_streams_runtime);
	
	/* ------------------------ END GPU Parallel Benchmarking ------------------------ */
}