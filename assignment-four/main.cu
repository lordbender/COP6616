#include "hip/hip_runtime.h"
#include <stdio.h>
#include "main_cuda.h"

#define N 16

// Device code
__global__ void VecSquare(float* A, float* C, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size)
        C[i] = A[i] * A[i];
}
            
// Host code
int main()
{
	size_t size = N;

	printf("Starting on size %d!!!\n", N);

    float* host_A = (float*)malloc(N);
    float* host_C = (float*)malloc(N);

	for (int i = 0; i < N; i++)
    {
		host_A[i] = (float)(i + i);
		host_C[i] = 0.0;
	}

	printf("Working Arrays Created\n");

    float* device_A;
	hipError_t hipMalloc(void** &device_A, size);
	printf("Cuda device_A Memory Allocated\n");

	// float* device_C;
	// hipMalloc((void**)&device_C, N);
	// printf("Cuda device_C Memory Allocated\n");

	// printf("Cuda Memory Allocated\n");

    // // Copy vector from host memory to device memory
    // hipMemcpy(device_A, host_A, N, hipMemcpyHostToDevice);

	// printf("Cuda Data Copy Completed\n");

    // // Invoke kernel
    // int threadsPerBlock = 256;
    // int blocksPerGrid =
	// 		(N + threadsPerBlock - 1) / threadsPerBlock;
			
    // VecSquare<<<blocksPerGrid, threadsPerBlock>>>(device_A, device_C, N);

	// hipPeekAtLastError();
	// hipDeviceSynchronize();

	// hipMemcpy(host_C, device_C, N, hipMemcpyDeviceToHost);
	
    // Free device memory
    hipFree(device_A);
	// hipFree(device_C);
	
 	// for (int i = 0; i < N; i++){
	// 	printf("\tCool Story %f\n", host_C[i]);
	// }

	free(host_A);
	free(host_C);
	
	printf("Done!!!\n");
}