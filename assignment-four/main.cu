#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.cuh"
           
int main(int argc, char *argv[])
{
  	srand(time(0));

 	int size = atoi(argv[1]);

    /* ------------------------ End CPU Sequential Benchmarking ------------------------ */
	
	double cpu_runtime = quicksort_cpu(size);
	printf("\n\tCPU O(n*log(n)): Completed %d numbers in %f seconds!!!\n", size, cpu_runtime);

	/* ------------------------ End CPU Sequential Benchmarking ------------------------ */

	
	/* ------------------------ Begin GPU Parallel Benchmarking Test Case ------------------------ */

	double gpu_runtime_n_operations = square_vector_gpu(size);
	printf("\tGPU O(n): Completed %d numbers in %f seconds!!!\n\n", size, gpu_runtime_n_operations);
	
	/* ------------------------ END GPU Parallel Benchmarking ------------------------ */

	/* ------------------------ Begin GPU Parallel Benchmarking ------------------------ */

	double gpu_runtime = quicksort_gpu(size);
	printf("\tGPU O(n*log(n)): Completed %d numbers in %f seconds!!!\n\n", size, gpu_runtime);
	
	/* ------------------------ END GPU Parallel Benchmarking ------------------------ */
}