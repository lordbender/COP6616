#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.cuh"
           
int main(int argc, char *argv[])
{
  	srand(time(0));

 	int size = atoi(argv[1]);

    /* ------------------------ End CPU Sequential Benchmarking ------------------------ */
	
	double cpu_runtime = quicksort_cpu(size);
	printf("\n\tCPU: Completed %d numbers in %f seconds!!!\n", size, cpu_runtime);

	/* ------------------------ End CPU Sequential Benchmarking ------------------------ */

	/* ------------------------ Begin GPU Parallel Benchmarking ------------------------ */

	double gpu_runtime = square_vector_gpu(size);
	printf("\tGPU: Completed %d numbers in %f seconds!!!\n\n", size, gpu_runtime);
	
	/* ------------------------ END GPU Parallel Benchmarking ------------------------ */
}