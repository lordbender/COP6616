#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "main_cuda.h"

static const int BLOCK_SIZE = 256;
static const int N = 10;

// Device code
__global__ void vecSquare(int* a, int* c, int n)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
        c[id] = a[id] * a[id];
}
            
// Host code
int main()
{
	printf("Starting on size %d!!!\n", N);

    int* host_A = (int*)malloc(N * sizeof(int*));
    int* host_C = (int*)malloc(N* sizeof(int*));

	for (int i = 0; i < N; i++)
    {
		host_A[i] = i + i;
		host_C[i] = 0.0;
	}

    int* device_A;
	gpuErrchk(hipMalloc((void**) &device_A, sizeof(int) * N));
	gpuErrchk(hipGetLastError());

	int* device_C;
	gpuErrchk(hipMalloc((void**) &device_C, sizeof(int) * N));
	gpuErrchk(hipGetLastError());
   
    gpuErrchk(hipMemcpy(device_A, host_A, N, hipMemcpyHostToDevice));
	gpuErrchk(hipGetLastError());

    int grid = ceil(N * 1.0 / BLOCK_SIZE);
    vecSquare<<<grid, BLOCK_SIZE>>>(device_A, device_C, N);
	hipDeviceSynchronize();
	gpuErrchk(hipGetLastError());

	hipMemcpy(host_C, device_C, sizeof(int) * N, hipMemcpyDeviceToHost);

    hipFree(device_A);
	hipFree(device_C);
	
 	for (int i = 0; i < N; i++) {
		printf("\tCool Story %d\n", host_C[i]);
	}

	free(host_A);
	free(host_C);
	
	hipDeviceReset();

	printf("Done!!!\n\n");
}