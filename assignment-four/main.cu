#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "main_cuda.h"

static const int BLOCK_SIZE = 256;
static const int N = 10;

// Device code
__global__ void vecSquare(float* a, float* c, int n)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
        c[id] = a[id] * a[id];
}
            
// Host code
int main()
{
	printf("Starting on size %d!!!\n", N);

    float* host_A = (float*)malloc(N * sizeof(float*));
    float* host_C = (float*)malloc(N* sizeof(float*));

	for (int i = 0; i < N; i++)
    {
		host_A[i] = (float)(i + i);
		host_C[i] = 0.0;
	}

    float* device_A;
	gpuErrchk(hipMalloc((void**) &device_A, sizeof(float) * N));
	gpuErrchk(hipGetLastError());

	float* device_C;
	gpuErrchk(hipMalloc((void**) &device_C, sizeof(float) * N));
	gpuErrchk(hipGetLastError());
   
    gpuErrchk(hipMemcpy(device_A, host_A, N, hipMemcpyHostToDevice));
	gpuErrchk(hipGetLastError());

    int grid = ceil(N * 1.0 / BLOCK_SIZE);
    vecSquare<<<grid, BLOCK_SIZE>>>(device_A, device_C, N);
	hipDeviceSynchronize();
	gpuErrchk(hipGetLastError());

	hipMemcpy(host_C, device_C, sizeof(float) * N, hipMemcpyDeviceToHost);

    hipFree(device_A);
	hipFree(device_C);
	
 	for (int i = 0; i < N; i++) {
		printf("\tCool Story %f\n", host_C[i]);
	}

	free(host_A);
	free(host_C);
	
	hipDeviceReset();

	printf("Done!!!\n\n");
}