#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.cuh"
           
int main(int argc, char *argv[])
{
  	srand(time(0));

 	int size = atoi(argv[1]);

    /* ------------------------ End CPU Sequential Benchmarking ------------------------ */
	
	double cpu_runtime = quickSort_cpu(size);
	printf("CPU: Completed %d numbers in %f seconds!!!\n\n", size, cpu_runtime);

	/* ------------------------ End CPU Sequential Benchmarking ------------------------ */

	/* ------------------------ Begin GPU Parallel Benchmarking ------------------------ */

	double gpu_runtime = quickSort_gpu(size);
	printf("GPU: Completed %d numbers in %f seconds!!!\n\n", size, gpu_runtime);
	
	/* ------------------------ END GPU Parallel Benchmarking ------------------------ */
}