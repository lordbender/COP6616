#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.cuh"
           
int main(int argc, char *argv[])
{
  	srand(time(0));

 	int size = atoi(argv[1]);

    /* ------------------------ End CPU Sequential Benchmarking ------------------------ */
	
	double cpu_runtime = quickSort_cpu(size);
	printf("\n\tCPU: Completed %d numbers in %f seconds!!!\n", size, cpu_runtime);

	/* ------------------------ End CPU Sequential Benchmarking ------------------------ */

	/* ------------------------ Begin GPU Parallel Benchmarking ------------------------ */

	double gpu_runtime = quickSort_gpu(size);
	printf("\tGPU: Completed %d numbers in %f seconds!!!\n\n", size, gpu_runtime);
	
	/* ------------------------ END GPU Parallel Benchmarking ------------------------ */
}