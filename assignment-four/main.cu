#include "hip/hip_runtime.h"
#include <stdio.h>
#include "main_cuda.h"

#define N 1000

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Device code
__global__ void VecSquare(float* A, float* C, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size)
        C[i] = A[i] * A[i];
}
            
// Host code
int main()
{

	printf("Starting on size %d!!!\n", N);

    float* h_A = (float*)malloc(N);
    float* h_C = (float*)malloc(N);

	for (int i = 0; i < N; i++)
    {
		h_A[i] = (float)(i + i);
		h_C[i] = 0.0;
	}
	printf("Working Arrays Created\n");

    float* d_A;
	gpuErrchk(hipMalloc(&d_A, N));
	
	float* d_C;
	gpuErrchk(hipMalloc(&d_C, N));

	printf("Cuda Memory Allocated\n");

    // Copy vector from host memory to device memory
    gpuErrchk(hipMemcpy(d_A, h_A, N, hipMemcpyHostToDevice));

	printf("Cuda Data Copy Completed\n");

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =
			(N + threadsPerBlock - 1) / threadsPerBlock;
			
    VecSquare<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, N);

	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	hipMemcpy(h_C, d_C, N, hipMemcpyDeviceToHost);
	


    // Free device memory
    hipFree(d_A);
	hipFree(d_C);
	
 	for (int i = 0; i < N; i++){
		printf("\tCool Story %f\n", h_C[i]);
	}

	free(h_A);
	free(h_C);
	
	printf("Done!!!\n");
}