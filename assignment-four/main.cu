#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.h"

static const int BLOCK_SIZE = 256;

// Device code
__global__ void vecSquare(int* a, int* c, int n)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
        c[id] = a[id] * a[id];
}
            
// Host code
double time_calc(clock_t start, clock_t end)
{
    return ((double)(end - start)) / CLOCKS_PER_SEC;
}
int main(int argc, char *argv[])
{
  	srand(time(0));

 	int size = atoi(argv[1]);

	int *ha, *hc, *da, *dc;
	printf("Starting on size %d!!!\n", size);

    ha = new int[size];
    hc = new int[size];

	for (int i = 0; i < size; i++)
    {
		ha[i] = i + i;
		hc[i] = 0;
	}

	clock_t start = clock();

	gpuErrchk(hipMalloc((void**) &da, sizeof(int) * size));
	gpuErrchk(hipGetLastError());

	gpuErrchk(hipMalloc((void**) &dc, sizeof(int) * size));
	gpuErrchk(hipGetLastError());
   
    gpuErrchk(hipMemcpy(da, ha,  sizeof(int) * size, hipMemcpyHostToDevice));
	gpuErrchk(hipGetLastError());

    int grid = ceil(size * 1.0 / BLOCK_SIZE);
    vecSquare<<<grid, BLOCK_SIZE>>>(da, dc, size);
	hipDeviceSynchronize();
	gpuErrchk(hipGetLastError());

	hipMemcpy(hc, dc, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(da);
	hipFree(dc);
	
	clock_t end = clock();

 	for (int i = 0; i < size; i++) {
		printf("\tCool Story %d\n", hc[i]);
	}

	free(ha);
	free(hc);
	
	hipDeviceReset();

	printf("Completed in %f seconds!!!\n\n", time_calc(start, end));
}