#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "main_cuda.h"

static const int BLOCK_SIZE = 256;
static const int N = 10;

// Device code
__global__ void vecSquare(float* a, float* c, int n)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
        c[id] = a[id] * a[id];
}
            
// Host code
int main()
{
	printf("Starting on size %d!!!\n", N);

    float* host_A = (float*)malloc(N * sizeof(float*));
    float* host_C = (float*)malloc(N* sizeof(float*));

	for (int i = 0; i < N; i++)
    {
		host_A[i] = (float)(i + i);
		host_C[i] = 0.0;
	}

	printf("Working Arrays Created\n");

    float* device_A;
	gpuErrchk(hipMalloc((void**) &device_A, sizeof(float) * N));
	printf("Cuda device_A Memory Allocated\n");

	gpuErrchk(hipGetLastError());

	float* device_C;
	gpuErrchk(hipMalloc((void**) &device_C, sizeof(float) * N));
	printf("Cuda device_C Memory Allocated\n");

	gpuErrchk(hipGetLastError());

	printf("Cuda Memory Allocated\n");

    // Copy vector from host memory to device memory
    gpuErrchk(hipMemcpy(device_A, host_A, N, hipMemcpyHostToDevice));

	printf("Cuda Data Copy Completed\n");

    // Invoke kernel
    int grid = ceil(N * 1.0 / BLOCK_SIZE);
    vecSquare<<<grid, BLOCK_SIZE>>>(device_A, device_C, N);

	hipDeviceSynchronize();

	gpuErrchk(hipGetLastError());

	// Copy the result back from the device to the host.
	hipMemcpy(host_C, device_C, sizeof(float) * N, hipMemcpyDeviceToHost);
	
    // Free device memory
    hipFree(device_A);
	hipFree(device_C);
	
 	for (int i = 0; i < N; i++) {
		printf("\tCool Story %f\n", host_C[i]);
	}

	free(host_A);
	free(host_C);
	
	hipDeviceReset();

	printf("Done!!!\n\n");
}