#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.h"

static const int BLOCK_SIZE = 256;
static const int N = 16;

// Device code
__global__ void vecSquare(int* a, int* c, int n)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
        c[id] = a[id] * a[id];
}
            
// Host code
double time_calc(clock_t start, clock_t end)
{
    return ((double)(end - start)) / CLOCKS_PER_SEC;
}
int main(int argc, char *argv[])
{
  	srand(time(0));

 	int size = atoi(argv[1]);

	int *ha, *hc, *da, *dc;
	printf("Starting on size %d!!!\n", N);

    ha = new int[N];
    hc = new int[N];

	for (int i = 0; i < N; i++)
    {
		ha[i] = i + i;
		hc[i] = 0;
	}

	clock_t start = clock();

	gpuErrchk(hipMalloc((void**) &da, sizeof(int) * N));
	gpuErrchk(hipGetLastError());

	gpuErrchk(hipMalloc((void**) &dc, sizeof(int) * N));
	gpuErrchk(hipGetLastError());
   
    gpuErrchk(hipMemcpy(da, ha,  sizeof(int) * N, hipMemcpyHostToDevice));
	gpuErrchk(hipGetLastError());

    int grid = ceil(N * 1.0 / BLOCK_SIZE);
    vecSquare<<<grid, BLOCK_SIZE>>>(da, dc, N);
	hipDeviceSynchronize();
	gpuErrchk(hipGetLastError());

	hipMemcpy(hc, dc, sizeof(int) * N, hipMemcpyDeviceToHost);

    hipFree(da);
	hipFree(dc);
	
	clock_t end = clock();

 	for (int i = 0; i < N; i++) {
		printf("\tCool Story %d\n", hc[i]);
	}

	free(ha);
	free(hc);
	
	hipDeviceReset();

	printf("Completed in %f seconds!!!\n\n", time_calc(start, end));
}