#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <ratio>
#include <chrono>

#include "main_cuda.cuh"

using namespace std::chrono;
 
int main(int argc, char *argv[])
{
  	srand(time(0));

 	int size = atoi(argv[1]);

	duration<double> cpu_radix_runtime = radixsort_cpu(size);
	printf("\n\n\tCPU O(n*log(n)) Sequential Radix: Completed %d numbers in %f seconds!!!\n", size, cpu_radix_runtime.count());
	   
	// duration<double> gpu_radix_runtime = radixsort_gpu(size);
	// printf("\tGPU O(n*log(n)) Streamed Radix: Completed %d numbers in %f seconds!!!\n\n", size, gpu_radix_runtime.count());

	duration<double> cpu_runtime = quicksort_cpu(size);
	printf("\tCPU O(n*log(n)) Sequential Quicksort: Completed %d numbers in %f seconds!!!\n", size, cpu_runtime.count());

	duration<double> cpu_threads_runtime = quicksort_cpu_threads(size);
	printf("\tCPU O(n*log(n)) Threaded Quicksort: Completed %d numbers in %f seconds!!!\n", size, cpu_threads_runtime.count());
	
	duration<double> gpu_streams_runtime = quicksort_gpu_streams(size);
	printf("\tGPU O(n*log(n)) Streamed: Completed %d numbers in %f seconds!!!\n\n", size, gpu_streams_runtime.count());
}