#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "main_cuda.cuh"

static const int BLOCK_SIZE = 256;

// Device code
__global__ void vecSquare(int* a, int* c, int n)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
        c[id] = a[id] * a[id];
}
            
int main(int argc, char *argv[])
{
  	srand(time(0));

 	int size = atoi(argv[1]);

	int *ha, *ha_seq, *hc, *da, *dc;
	printf("Starting on size %d!!!\n", size);

    ha = (int*)malloc(sizeof(int) * size);
    hc = (int*)malloc(sizeof(int) * size);
    ha_seq = (int*)malloc(sizeof(int) * size);

	// Fill the tet arrays
	for (int i = 0; i < size; i++)
    {
		int rand_num = rand();
		ha[i] = rand_num;
		ha_seq[i] = rand_num;
		hc[i] = 0;
	}

	/* ------------------------ Begin CPU Sequential Benchmarking ------------------------ */

	// Sequential For Benchmarking
	clock_t start_seq = clock();
	quickSort(ha_seq, 0, size - 1);
	clock_t end_seq = clock();

	// Testing that sort is working, keep commented out on large values of N (say N > 1000)
	// for (int i = 0; i < size; i++) {
    // 	printf("\t %d\n", ha_seq[i]);
	// }
	
	printf("CPU: Completed %d numbers in %f seconds!!!\n\n", size, time_calc(start_seq, end_seq));

	/* ------------------------ End CPU Sequential Benchmarking ------------------------ */

	/* ------------------------ Begin GPU Parallel Benchmarking ------------------------ */
	clock_t start = clock();

	gpuErrchk(hipMalloc((void**) &da, sizeof(int) * size));
	gpuErrchk(hipGetLastError());

	gpuErrchk(hipMalloc((void**) &dc, sizeof(int) * size));
	gpuErrchk(hipGetLastError());
   
    gpuErrchk(hipMemcpy(da, ha,  sizeof(int) * size, hipMemcpyHostToDevice));
	gpuErrchk(hipGetLastError());

    int grid = ceil(size * 1.0 / BLOCK_SIZE);
    vecSquare<<<grid, BLOCK_SIZE>>>(da, dc, size);
	hipDeviceSynchronize();
	gpuErrchk(hipGetLastError());

	hipMemcpy(hc, dc, sizeof(int) * size, hipMemcpyDeviceToHost);

    hipFree(da);
	hipFree(dc);
	
	clock_t end = clock();

	free(ha);
	free(hc);
	
	hipDeviceReset();

	// Testing that sort is working, keep commented out on large values of N (say N > 1000)
	// for (int i = 0; i < size; i++) {
    // 	printf("\t %d\n", hc[i]);
	// }
	
	printf("GPU: Completed %d numbers in %f seconds!!!\n\n", size, time_calc(start, end));
	
	/* ------------------------ END GPU Parallel Benchmarking ------------------------ */
}