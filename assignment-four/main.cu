#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "main_cuda.h"

static const int BLOCK_SIZE = 256;
static const int N = 16;

// Device code
__global__ void vecSquare(int* a, int* c, int n)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
        c[id] = a[id] * a[id];
}
            
// Host code
int main()
{
	int *ha, *hc, *da, *dc;
	printf("Starting on size %d!!!\n", N);

    ha = new int[N];
    hc = new int[N];

	for (int i = 0; i < N; i++)
    {
		ha[i] = i + i;
		hc[i] = 0;
	}

	gpuErrchk(hipMalloc((void**) &da, sizeof(int) * N));
	gpuErrchk(hipGetLastError());

	gpuErrchk(hipMalloc((void**) &dc, sizeof(int) * N));
	gpuErrchk(hipGetLastError());
   
    gpuErrchk(hipMemcpy(da, ha,  sizeof(int) * N, hipMemcpyHostToDevice));
	gpuErrchk(hipGetLastError());

    int grid = ceil(N * 1.0 / BLOCK_SIZE);
    vecSquare<<<grid, BLOCK_SIZE>>>(da, dc, N);
	hipDeviceSynchronize();
	gpuErrchk(hipGetLastError());

	hipMemcpy(ha, dc, sizeof(int) * N, hipMemcpyDeviceToHost);

    hipFree(da);
	hipFree(dc);
	
 	for (int i = 0; i < N; i++) {
		printf("\tCool Story %d\n", hc[i]);
	}

	free(ha);
	free(hc);
	
	hipDeviceReset();

	printf("Done!!!\n\n");
}