
#include <hip/hip_runtime.h>
// Device code
__global__ void VecAdd(long* A, long* B, long* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}
            
// Host code
int main()
{
    int N = 1000;
    size_t size = N * sizeof(long);

    // Allocate input vectors h_A and h_B in host memory
    long* h_A = (long*)malloc(size);
    long* h_B = (long*)malloc(size);
    long* h_C = (long*)malloc(size);

	for (int i = 0; i < size; ++i)
    {
		h_A[i] = i + i;
		h_B[i] = i - i;
		h_C[i] = 0;
	}

    // Allocate vectors in device memory
    long* d_A;
    hipMalloc(&d_A, size);
    long* d_B;
    hipMalloc(&d_B, size);
    long* d_C;
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =
            (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
 
}